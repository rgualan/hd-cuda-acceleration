#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "spt1.h"
#include "spt2.h"
#include "multiGPU.h"

__global__ void horizontal_diffusion_kernel(int ids, int ide, int jds, int jde, int kds, int kde,
		int ims, int ime, int jms, int jme, int kms, int kme, int its, int ite, int jts, int jte,
		int kts, int kte, int cf_specified, int cf_nested, int cf_open_xs, int cf_open_xe,
		int cf_open_ys, int cf_open_ye, int cf_periodic_x, int cf_polar, char name, float *field,
		float *tendency, float *mu, float *msfux, float *msfuy, float *msfvx, float *msfvx_inv,
		float *msfvy, float *msftx, float *msfty, float khdif, float *xkmhd, float rdx, float rdy,
		int sub_js, int sub_je, int data_js, int data_je) {

	// Dimensions of the 3d grid
	const int ix = ime - ims + 1;
	const int jx = data_je - data_js + 1;
	const int kx = kme - kms + 1;

	// Local variables
	int i, j, k, ktf;
	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;

	if (cf_specified || cf_nested) {
		specified = 1;
	}

	ktf = MIN(kte,kde-1);

	// Thread coordinates
	i = bi * bx + ti;
	j = bj * by + tj;

	if (name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = MIN(jte,jde-1);

		if (cf_open_xs || specified)
			i_start = MAX(ids+1,its);
		if (cf_open_xe || specified)
			i_end = MIN(ide-1,ite);
		if (cf_open_ys || specified)
			j_start = MAX(jds+1,jts);
		if (cf_open_ye || specified)
			j_end = MIN(jde-2,jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = ite;

		j_start = MAX(j_start, sub_js); // In Fortran type index
		j_end = MIN(j_end, sub_je); // In Fortran type index

		// Check bounds before calculations!
		if (!(i >= I(i_start) && i <= I(i_end) && j >= j_start - data_js && j <= j_end - data_js)) {
			return;
		}

		for (k = K(kts); k <= K(ktf); k++) {

			// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
			// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

			mkrdxm = (msftx[Q2(ti - 1, tj)] / msfty[Q2(ti - 1, tj)]) * mu[Q2(ti - 1, tj)]
					* xkmhd[Q3(ti - 1, k, tj)] * (rdx);
			mkrdxp = (msftx[Q2(ti, tj)] / msfty[Q2(ti, tj)]) * mu[Q2(ti, tj)] * xkmhd[Q3(ti, k, tj)]
					* (rdx);
			mrdx = msfux[Q2(ti, tj)] * msfuy[Q2(ti, tj)] * (rdx);
			mkrdym = ((msfuy[Q2(ti, tj)] + msfuy[Q2(ti, tj - 1)])
					/ (msfux[Q2(ti, tj)] + msfux[Q2(ti, tj - 1)])) * 0.25
					* (mu[Q2(ti, tj)] + mu[Q2(ti, tj - 1)] + mu[Q2(ti - 1, tj - 1)]
							+ mu[Q2(ti - 1, tj)]) * 0.25
					* (xkmhd[Q3(ti, k, tj)] + xkmhd[Q3(ti, k, tj - 1)]
							+ xkmhd[Q3(ti - 1, k, tj - 1)] + xkmhd[Q3(ti - 1, k, tj)]) * (rdy);
			mkrdyp = ((msfuy[Q2(ti, tj)] + msfuy[Q2(ti, tj + 1)])
					/ (msfux[Q2(ti, tj)] + msfux[Q2(ti, tj + 1)])) * 0.25
					* (mu[Q2(ti, tj)] + mu[Q2(ti, tj + 1)] + mu[Q2(ti - 1, tj + 1)]
							+ mu[Q2(ti - 1, tj)]) * 0.25
					* (xkmhd[Q3(ti, k, tj)] + xkmhd[Q3(ti, k, tj + 1)]
							+ xkmhd[Q3(ti - 1, k, tj + 1)] + xkmhd[Q3(ti - 1, k, tj)]) * (rdy);

			// need to do four-corners (t) for diffusion coefficient as there are
			// no values at u,v points
			// msfuy - has to be y as part of d/dY
			//         has to be u as we're at a u point
			mrdy = msfux[Q2(ti, tj)] * msfuy[Q2(ti, tj)] * (rdy);

			// correctly averaged version of rho~ * m^2 *
			//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]

			tendency[Q3(ti, k, tj)] = tendency[Q3(ti, k, tj)]
					+ (mrdx
							* (mkrdxp * (field[Q3(ti + 1, k, tj)] - field[Q3(ti, k, tj)])
									- mkrdxm * (field[Q3(ti, k, tj)] - field[Q3(ti - 1, k, tj)]))
							+ mrdy
									* (mkrdyp * (field[Q3(ti, k, tj + 1)] - field[Q3(ti, k, tj)])
											- mkrdym
													* (field[Q3(ti, k, tj)]
															- field[Q3(ti, k, tj - 1)])));
		}
	}

}
