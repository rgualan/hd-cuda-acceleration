#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h> // CUDA runtime
#include <hip/hip_runtime_api.h>  // Utilities to work with CUDA
#include "spt1.h"
#include "multiGPU.h"
//#include "util.h"
#include "hd_block_size.h"

// Moving data
// To host (page-lock/pinned)
//#define TOHOST(A,d,s) checkCudaErrors(hipHostMalloc((void**)&plan[i].A##_h,(d)*sizeof(float)));memcpy(&plan[i].A##_h[0], &A[(s)], (d)*sizeof(float))
#define TOHOST(A,d,s) hipHostMalloc((void**)&plan[i].A##_h,(d)*sizeof(float));memcpy(&plan[i].A##_h[0], &A[(s)], (d)*sizeof(float))
#define TOHOST3(A) TOHOST(A,plan[i].d3,plan[i].s3)
#define TOHOST2(A) TOHOST(A,plan[i].d2,plan[i].s2)

// To device, from pinned memory
//#define TODEV(A,d) printf("TODEV:%d\n",d);checkCudaErrors(hipMalloc((void**)&plan[i].A##_d,(d)*sizeof(float)));printf("Pasa hipMalloc\n");checkCudaErrors(hipMemcpyAsync(plan[i].A##_d, plan[i].A##_h, (d)*sizeof(float),hipMemcpyHostToDevice, plan[i].stream));printf("Pasa hipMemcpyAsync\n");
#define TODEV(A,d) hipMalloc((void**)&plan[i].A##_d,(d)*sizeof(float));hipMemcpyAsync(plan[i].A##_d, plan[i].A##_h, (d)*sizeof(float),hipMemcpyHostToDevice, plan[i].stream)
#define TODEV3(A) TODEV(A,plan[i].d3)
#define TODEV2(A) TODEV(A,plan[i].d2)

// To host, from device, through pinned memory
//#define FROMDEV(A,d1,d2,s1,s2) checkCudaErrors(hipMemcpyAsync(plan[i].A##_h,plan[i].A##_d,(d1)*sizeof(float),hipMemcpyDeviceToHost,plan[i].stream));memcpy(&A[s1], &plan[i].A##_h[(s2)], (d2)*sizeof(float));
//#define FROMDEV(A,d1,d2,s1,s2) hipMemcpyAsync(&plan[i].A##_h, &plan[i].A##_d, (d1) *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream);memcpy(&A[s1], &plan[i].A##_h[(s2)], (d2)*sizeof(float));
//#define FROMDEV3(A) FROMDEV(A,plan[i].d3,plan[i].d3_,plan[i].s3_,plan[i].delta)
//#define FROMDEV2(A) FROMDEV(A,plan[i].d2,plan[i].d2_,plan[i].s2_,plan[i].delta)

#define CLNUP(A) hipFree(plan[i].A##_d)

// Max number of GPUs used
#define MAX_GPU_COUNT 32

/*float * allocAndCopyCuda(float * var, int size, startIndex) {
 float *var2;
 hipMalloc((void**) &var2, (size) * sizeof(float));
 hipMemcpy(var2, &var[startIndex], (size) * sizeof(float),
 hipMemcpyHostToDevice);

 return var2;
 }*/
extern __global__ void horizontal_diffusion_kernel(int ids, int ide, int jds, int jde, int kds,
		int kde, int ims, int ime, int jms, int jme, int kms, int kme, int its, int ite, int jts,
		int jte, int kts, int kte, int cf_specified, int cf_nested, int cf_open_xs, int cf_open_xe,
		int cf_open_ys, int cf_open_ye, int cf_periodic_x, int cf_polar, char name, float *field,
		float *tendency, float *mu, float *msfux, float *msfuy, float *msfvx, float *msfvx_inv,
		float *msfvy, float *msftx, float *msfty, float khdif, float *xkmhd, float rdx, float rdy,
		int sub_js, int sub_je, int data_js, int data_je);

extern "C" {

int horizontal_diffusion_host(int *ids, int *ide, int *jds, int *jde, int *kds, int *kde, int *ims,
		int *ime, int *jms, int *jme, int *kms, int *kme, int *its, int *ite, int *jts, int *jte,
		int *kts, int *kte, int *cf_specified, int *cf_nested, int *cf_open_xs, int *cf_open_xe,
		int *cf_open_ys, int *cf_open_ye, int *cf_periodic_x, int *cf_polar, char *name,
		float *field, float *tendency, float *mu, float *msfux, float *msfuy, float *msfvx,
		float *msfvx_inv, float *msfvy, float *msftx, float *msfty, float *khdif, float *xkmhd,
		float *rdx, float *rdy) {

	int i; // Index
	int GPU_N; // Number of GPUs

	// Timing data
	hipEvent_t tS0, tE0;
	hipEventCreate(&tS0);
	hipEventCreate(&tE0);
	float timer = 0.0f;

	// Original dimensions (Not considering sub-division in j dimension) (For indexing macros)
	const int ix = *ime - *ims + 1;
	const int kx = *kme - *kms + 1;

	printf("Input name: %c\n", *name);
	printf("Tendency first element: %f\n", tendency[P3(6,0,6)]);

	printf("Dimensions:\n");
	printf("ids,ide,jds,jde,kds,kde: %4d,%4d,%4d,%4d,%4d,%4d\n", *ids, *ide, *jds, *jde, *kds,
			*kde);
	printf("ims,ime,jms,jme,kms,kme: %4d,%4d,%4d,%4d,%4d,%4d\n", *ims, *ime, *jms, *jme, *kms,
			*kme);
	printf("its,ite,jts,jte,kts,kte: %4d,%4d,%4d,%4d,%4d,%4d\n", *its, *ite, *jts, *jte, *kts,
			*kte);

	// Solver configuration
	// This structure has the basic I/O data for each calculation of a kernel
	TGPUplan plan[MAX_GPU_COUNT];

	// Get the number of available devices in the node
	// The limit is MAX_GPU_COUNT
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	if (GPU_N > MAX_GPU_COUNT) {
		GPU_N = MAX_GPU_COUNT;
	}
	GPU_N = 2; //TODO
	printf("CUDA-capable device count: %i\n", GPU_N);

	// Subdivide workload magnitude
	printf("Subdividing workload...\n");
	for (i = 0; i < GPU_N; i++) {
		plan[i].sub_dim_j = (*jme - *jms + 1) / GPU_N;
	}
	//Take into account "odd" data sizes
	for (i = 0; i < (*jme - *jms + 1) % GPU_N; i++) {
		plan[i].sub_dim_j++;
	}

	// Define indices for processing and for sub-data loading
	for (i = 0; i < GPU_N; i++) {
		// Sub-region processing indices
		if (i == 0) {
			plan[i].sub_js = *jms;
			plan[i].sub_je = plan[i].sub_js + plan[i].sub_dim_j - 1;
		} else {
			plan[i].sub_js = plan[i - 1].sub_je + 1;
			plan[i].sub_je = plan[i].sub_js + plan[i].sub_dim_j - 1;
		}
		printf("Subdivision of proc (GPU %d) in j dim: (%4d,%4d) (%d)\n", i, plan[i].sub_js,
				plan[i].sub_je, plan[i].sub_dim_j);

		// Sub-region data indices
		if (plan[i].sub_js - 1 < (*ims))
			plan[i].data_js = (*ims);
		else
			plan[i].data_js = plan[i].sub_js - 1;

		if (plan[i].sub_je + 1 > (*jme))
			plan[i].data_je = (*jme);
		else
			plan[i].data_je = plan[i].sub_je + 1;
		printf("Subdivision of data (GPU %d) in j dim: (%4d,%4d) (%d)\n", i, plan[i].data_js,
				plan[i].data_je, (plan[i].data_je - plan[i].data_js + 1));
	}

	//startTimer(1);
	hipEventRecord(tS0, NULL);

	//Create streams for issuing GPU command asynchronously and allocate memory
	//(GPU and System page-locked/pinned)
	for (i = 0; i < GPU_N; i++) {
		//checkCudaErrors(hipSetDevice(i)); //TODO
		checkCudaErrors(hipSetDevice(0));
		checkCudaErrors(hipStreamCreate(&plan[i].stream));
		// Allocate memory
		// Indices and dimensions for transference
		plan[i].s3 = J(plan[i].data_js) * ((*ime - *ims + 1) * (*kme - *kms + 1));
		plan[i].s2 = J(plan[i].data_js) * ((*ime - *ims + 1));
		plan[i].d3 = (*ime - *ims + 1) * (*kme - *kms + 1)
				* (plan[i].data_je - plan[i].data_js + 1);
		plan[i].d2 = (*ime - *ims + 1) * (plan[i].data_je - plan[i].data_js + 1);

		//Transfer macros to host
		TOHOST3(field);
		TOHOST3(tendency);
		TOHOST3(xkmhd);
		TOHOST2(mu);
		TOHOST2(msfux);
		TOHOST2(msfuy);
		TOHOST2(msfvx);
		TOHOST2(msfvx_inv);
		TOHOST2(msfvy);
		TOHOST2(msftx);
		TOHOST2(msfty);

		// Transfer macros to device
		TODEV3(field);
		TODEV3(tendency);
		TODEV3(xkmhd);
		TODEV2(mu);
		TODEV2(msfux);
		TODEV2(msfuy);
		TODEV2(msfvx);
		TODEV2(msfvx_inv);
		TODEV2(msfvy);
		TODEV2(msftx);
		TODEV2(msfty);

		printf("Computing with GPU %d/%d...\n", i, GPU_N);

		int remx, remy;
		remx = (*ime - *ims + 1) % XXX != 0 ? 1 : 0;
		remy = (plan[i].sub_je - plan[i].sub_js + 1) % YYY != 0 ? 1 : 0;
		dim3 dimBlock(XXX, YYY);
		dim3 dimGrid(((*ime - *ims + 1) / XXX) + remx,
				((plan[i].sub_je - plan[i].sub_js + 1) / YYY) + remy);

		printf("Call to kernel: block dims (%4d,%4d)\n", dimBlock.x, dimBlock.y);
		printf("Call to kernel: grid  dims (%4d,%4d)\n", dimGrid.x, dimGrid.y);

		horizontal_diffusion_kernel<<<dimGrid, dimBlock, 0, plan[i].stream>>>(
				*ids, *ide, *jds, *jde, *kds, *kde, *ims, *ime, *jms,
				*jme, *kms, *kme, *its, *ite, *jts, *jte, *kts, *kte, *cf_specified,
				*cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye,
				*cf_periodic_x, *cf_polar, *name, plan[i].field_d, plan[i].tendency_d, plan[i].mu_d,
				plan[i].msfux_d, plan[i].msfuy_d, plan[i].msfvx_d, plan[i].msfvx_inv_d, plan[i].msfvy_d, plan[i].msftx_d,
				plan[i].msfty_d, *khdif, plan[i].xkmhd_d, *rdx, *rdy, plan[i].sub_js,
				plan[i].sub_je, plan[i].data_js, plan[i].data_je);
		getLastCudaError("<Kernel execution failed>");

		// Indices for returning data
		plan[i].s3_ = J(plan[i].sub_js) * ((*ime - *ims + 1) * (*kme - *kms + 1));
		plan[i].d3_ = (*ime - *ims + 1) * (*kme - *kms + 1) * (plan[i].sub_je - plan[i].sub_js + 1);
		plan[i].delta = (plan[i].sub_js - plan[i].data_js) * (*ime - *ims + 1) * (*kme - *kms + 1);

		//Read back GPU results to pinned memory
		hipMemcpyAsync(plan[i].tendency_h, plan[i].tendency_d, (plan[i].d3) * sizeof(float),
				hipMemcpyDeviceToHost, plan[i].stream);
	}

	//Synchronize GPU and shutdown
	for (i = 0; i < GPU_N; i++) {
		//Set device
		//checkCudaErrors(hipSetDevice(i));
		//hipSetDevice(i);
		hipSetDevice(0); //TODO

		//Wait for all operations to finish
		hipStreamSynchronize(plan[i].stream);

		// Read back GPU result to Host memory from pinned memory
		memcpy(&tendency[plan[i].s3_], &plan[i].tendency_h[(plan[i].delta)],
				(plan[i].d3_) * sizeof(float));

		// Clean memory
		CLNUP(field);
		CLNUP(tendency);
		CLNUP(xkmhd);
		CLNUP(mu);
		CLNUP(msfux);
		CLNUP(msfuy);
		CLNUP(msfvx);
		CLNUP(msfvx_inv);
		CLNUP(msfvy);
		CLNUP(msftx);
		CLNUP(msfty);

		//Shut down this GPU
		//checkCudaErrors(hipStreamDestroy(plan[i].stream));
		hipStreamDestroy(plan[i].stream);
	}

	hipEventRecord(tE0, NULL);
	hipEventSynchronize(tE0);
	hipEventElapsedTime(&timer, tS0, tE0);
	printf("Runtime of kernel over %d GPUS using streams: %.3f msec\n", GPU_N, timer);

	return 0;
}

}
