#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#include "hd_block_size.h"
#include "spt2.h"

//__constant__ float rdx_[1];
//__constant__ float rdy_[1];

__global__ void horizontal_diffusion_gpu(const int ids, const int ide,
		const int jds, const int jde, const int kds, const int kde,
		const int ims, const int ime, const int jms, const int jme,
		const int kms, const int kme, const int its, const int ite,
		const int jts, const int jte, const int kts, const int kte,
		const int cf_specified, const int cf_nested, const int cf_open_xs,
		const int cf_open_xe, const int cf_open_ys, const int cf_open_ye,
		const int cf_periodic_x, const int cf_polar, const char name,
		const float *field, float *tendency, const float *mu,
		const float *msfux, const float *msfuy, const float *msfvx,
		const float *msfvx_inv, const float *msfvy, const float *msftx,
		const float *msfty, const float khdif, const float *xkmhd,
		const float rdx, const float rdy) {

	// Local variables
	//int i, j, k, ktf;

	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;

	// Esto hace que aumente considerablemente el tiempo (+40%)
	// en lugar de disminuirlo

	if (cf_specified || cf_nested) {
		specified = 1;
	}

	const int ktf = min(kte, kde - 1);

	// Thread coordinates
	const int i = bi * bx + ti;
	const int j = bj * by + tj;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;

//  if (name == 'u') {
	i_start = its;
	i_end = ite;
	j_start = jts;
	j_end = min(jte, jde - 1);

	if (cf_open_xs || specified)
		i_start = max(ids + 1, its);
	if (cf_open_xe || specified)
		i_end = min(ide - 1, ite);
	if (cf_open_ys || specified)
		j_start = max(jds + 1, jts);
	if (cf_open_ye || specified)
		j_end = min(jde - 2, jte);
	if (cf_periodic_x)
		i_start = its;
	if (cf_periodic_x)
		i_end = ite;

	// Check bounds before calculations!
	if (!(i >= i_start + 4 && i <= i_end + 4 && j >= j_start + 4
			&& j <= j_end + 4 && k <= ktf - 1)) {
		return;
	}

	// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
	// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

	mkrdxm = (msftx[P2(ti-1,tj)] / msfty[P2(ti-1,tj)]) * mu[P2(ti-1,tj)]
			* xkmhd[P3(ti-1,k,tj)] * rdx;
	mkrdxp = (msftx[P2(ti,tj)] / msfty[P2(ti,tj)]) * mu[P2(ti,tj)]
			* xkmhd[P3(ti,k,tj)] * rdx;
	mrdx = msfux[P2(ti,tj)] * msfuy[P2(ti,tj)] * rdx;

	mkrdym = ((msfuy[P2(ti,tj)] + msfuy[P2(ti,tj-1)])
			/ (msfux[P2(ti,tj)] + msfux[P2(ti,tj-1)])) * 0.25
			* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti-1,tj-1)]
					+ mu[P2(ti-1,tj)]) * 0.25
			* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
					+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)]) * rdy;
	mkrdyp = ((msfuy[P2(ti,tj)] + msfuy[P2(ti,tj+1)])
			/ (msfux[P2(ti,tj)] + msfux[P2(ti,tj+1)])) * 0.25
			* (mu[P2(ti,tj)] + mu[P2(ti,tj+1)] + mu[P2(ti-1,tj+1)]
					+ mu[P2(ti-1,tj)]) * 0.25
			* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj+1)]
					+ xkmhd[P3(ti-1,k,tj+1)] + xkmhd[P3(ti-1,k,tj)]) * rdy;

	// need to do four-corners (t) for diffusion coefficient as there are
	// no values at u,v points
	// msfuy - has to be y as part of d/dY
	//         has to be u as we're at a u point
	mrdy = msfux[P2(ti,tj)] * msfuy[P2(ti,tj)] * rdy;

	// correctly averaged version of rho~ * m^2 *
	//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]
	/*tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
	 mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
	 -mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
	 +mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
	 -mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));*/
	tendency[P3(ti,k,tj)] =
			tendency[P3(ti,k,tj)]
					+ (mrdx
							* (mkrdxp
									* (field[P3(ti+1,k,tj)] - field[P3(ti,k,tj)])
									- mkrdxm
											* (field[P3(ti,k,tj)]
													- field[P3(ti-1,k,tj)]))
							+ mrdy
									* (mkrdyp
											* (field[P3(ti,k,tj+1)]
													- field[P3(ti,k,tj)])
											- mkrdym
													* (field[P3(ti,k,tj)]
															- field[P3(ti,k,tj-1)])));
	/*	} else if (name == 'v') {

	 i_start = its;
	 i_end = min(ite, ide - 1);
	 j_start = jts;
	 j_end = jte;

	 if (cf_open_xs || specified)
	 i_start = max(ids + 1, its);
	 if (cf_open_xe || specified)
	 i_end = min(ide - 2, ite);
	 if (cf_open_ys || specified)
	 j_start = max(jds + 1, jts);
	 if (cf_open_ye || specified)
	 j_end = min(jde - 1, jte);
	 if (cf_periodic_x)
	 i_start = its;
	 if (cf_periodic_x)
	 i_end = min(ite, ide - 1);
	 if (cf_polar)
	 j_start = max(jds + 1, jts);
	 if (cf_polar)
	 j_end = min(jde - 1, jte);

	 // Check bounds before calculations!
	 if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
	 && j <= j_end - 1)) {
	 return;
	 }

	 for (k = kts - 1; k <= ktf - 1; k++) {

	 mkrdxm = ((msfvx[P2(ti,tj)] + msfvx[P2(ti-1,tj)])
	 / (msfvy[P2(ti,tj)] + msfvy[P2(ti-1,tj)])) * 0.25
	 * (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti-1,tj-1)]
	 + mu[P2(ti-1,tj)]) * 0.25
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
	 + xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
	 * rdx;
	 mkrdxp = ((msfvx[P2(ti,tj)] + msfvx[P2(ti+1,tj)])
	 / (msfvy[P2(ti,tj)] + msfvy[P2(ti+1,tj)])) * 0.25
	 * (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti+1,tj-1)]
	 + mu[P2(ti+1,tj)]) * 0.25
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
	 + xkmhd[P3(ti+1,k,tj-1)] + xkmhd[P3(ti+1,k,tj)])
	 * rdx;
	 mrdx = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdx;
	 mkrdym = (msfty[P2(ti,tj-1)] / msftx[P2(ti,tj-1)])
	 * xkmhd[P3(ti,k,tj-1)] * rdy;
	 mkrdyp = (msfty[P2(ti,tj)] / msftx[P2(ti,tj)]) * xkmhd[P3(ti,k,tj)]
	 * rdy;
	 mrdy = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdy;

	 tendency[P3(ti,k,tj)] =
	 tendency[P3(ti,k,tj)]
	 + (mrdx
	 * (mkrdxp
	 * (field[P3(ti+1,k,tj)]
	 - field[P3(ti,k,tj)])
	 - mkrdxm
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti-1,k,tj)]))
	 + mrdy
	 * (mkrdyp
	 * (field[P3(ti,k,tj+1)]
	 - field[P3(ti,k,tj)])
	 - mkrdym
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti,k,tj-1)])));
	 }
	 } else if (name == 'w') {

	 i_start = its;
	 i_end = min(ite, ide - 1);
	 j_start = jts;
	 j_end = min(jte, jde - 1);

	 if (cf_open_xs || specified)
	 i_start = max(ids + 1, its);
	 if (cf_open_xe || specified)
	 i_end = min(ide - 2, ite);
	 if (cf_open_ys || specified)
	 j_start = max(jds + 1, jts);
	 if (cf_open_ye || specified)
	 j_end = min(jde - 2, jte);
	 if (cf_periodic_x)
	 i_start = its;
	 if (cf_periodic_x)
	 i_end = min(ite, ide - 1);

	 // Check bounds before calculations!
	 if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
	 && j <= j_end - 1)) {
	 return;
	 }

	 for (k = kts - 1; k <= ktf - 1; k++) {

	 mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.25
	 * (mu[P2(ti,tj)] + mu[P2(ti-1,tj)] + mu[P2(ti,tj)]
	 + mu[P2(ti-1,tj)]) * 0.25
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]
	 + xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti-1,k-1,tj)])
	 * rdx;
	 mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.25
	 * (mu[P2(ti+1,tj)] + mu[P2(ti,tj)] + mu[P2(ti+1,tj)]
	 + mu[P2(ti,tj)]) * 0.25
	 * (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]
	 + xkmhd[P3(ti+1,k-1,tj)] + xkmhd[P3(ti,k-1,tj)])
	 * rdx;
	 mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
	 //         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*
	 mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.25
	 * (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti,tj)]
	 + mu[P2(ti,tj-1)]) * 0.25
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
	 + xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti,k-1,tj-1)])
	 * rdy;
	 //         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*
	 mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.25
	 * (mu[P2(ti,tj+1)] + mu[P2(ti,tj)] + mu[P2(ti,tj+1)]
	 + mu[P2(ti,tj)]) * 0.25
	 * (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]
	 + xkmhd[P3(ti,k-1,tj+1)] + xkmhd[P3(ti,k-1,tj)])
	 * rdy;
	 mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

	 tendency[P3(ti,k,tj)] =
	 tendency[P3(ti,k,tj)]
	 + (mrdx
	 * (mkrdxp
	 * (field[P3(ti+1,k,tj)]
	 - field[P3(ti,k,tj)])
	 - mkrdxm
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti-1,k,tj)]))
	 + mrdy
	 * (mkrdyp
	 * (field[P3(ti,k,tj+1)]
	 - field[P3(ti,k,tj)])
	 - mkrdym
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti,k,tj-1)])));
	 }
	 } else {

	 i_start = its;
	 i_end = min(ite, ide - 1);
	 j_start = jts;
	 j_end = min(jte, jde - 1);

	 if (cf_open_xs || specified)
	 i_start = max(ids + 1, its);
	 if (cf_open_xe || specified)
	 i_end = min(ide - 2, ite);
	 if (cf_open_ys || specified)
	 j_start = max(jds + 1, jts);
	 if (cf_open_ye || specified)
	 j_end = min(jde - 2, jte);
	 if (cf_periodic_x)
	 i_start = its;
	 if (cf_periodic_x)
	 i_end = min(ite, ide - 1);

	 for (k = kts; k <= ktf; k++) {

	 mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.5
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]) * 0.5
	 * (mu[P2(ti,tj)] + mu[P2(ti-1,tj)]) * rdx;
	 mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.5
	 * (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]) * 0.5
	 * (mu[P2(ti+1,tj)] + mu[P2(ti,tj)]) * rdx;
	 mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
	 //         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy
	 mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.5
	 * (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]) * 0.5
	 * (mu[P2(ti,tj)] + mu[P2(ti,tj-1)]) * rdy;
	 //         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy
	 mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.5
	 * (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]) * 0.5
	 * (mu[P2(ti,tj+1)] + mu[P2(ti,tj)]) * rdy;
	 mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

	 tendency[P3(ti,k,tj)] =
	 tendency[P3(ti,k,tj)]
	 + (mrdx
	 * (mkrdxp
	 * (field[P3(ti+1,k,tj)]
	 - field[P3(ti,k,tj)])
	 - mkrdxm
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti-1,k,tj)]))
	 + mrdy
	 * (mkrdyp
	 * (field[P3(ti,k,tj+1)]
	 - field[P3(ti,k,tj)])
	 - mkrdym
	 * (field[P3(ti,k,tj)]
	 - field[P3(ti,k,tj-1)])));
	 }
	 }
	 */
}
