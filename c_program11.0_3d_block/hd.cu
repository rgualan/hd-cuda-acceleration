#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hd_block_size.h"

#define TODEV(A,s) float *A##_d;hipMalloc((void**)&A##_d,((s))*sizeof(float));hipMemcpy(A##_d,A,(s)*sizeof(float),hipMemcpyHostToDevice);
#define FROMDEV(A,s) hipMemcpy(A,A##_d,(s)*sizeof(float),hipMemcpyDeviceToHost);
#define CLNUP(A) hipFree(A##_d)

#define TODEV3(A) TODEV(A,d3)
#define TODEV2(A) TODEV(A,d2)
#define FROMDEV3(A) FROMDEV(A,d3)
#define FROMDEV2(A) FROMDEV(A,d2)

//extern __constant__ float rdx_[1];
//extern __constant__ float rdy_[1];

__global__ void horizontal_diffusion_gpu(const int ids, const int ide,
		const int jds, const int jde, const int kds, const int kde,
		const int ims, const int ime, const int jms, const int jme,
		const int kms, const int kme, const int its, const int ite,
		const int jts, const int jte, const int kts, const int kte,
		const int cf_specified, const int cf_nested, const int cf_open_xs,
		const int cf_open_xe, const int cf_open_ys, const int cf_open_ye,
		const int cf_periodic_x, const int cf_polar, const char name,
		const float *field, float *tendency, const float *mu,
		const float *msfux, const float *msfuy, const float *msfvx,
		const float *msfvx_inv, const float *msfvy, const float *msftx,
		const float *msfty, const float khdif, const float *xkmhd,
		const float rdx, const float rdy);

extern "C" int gethostname(char * name, size_t len);

extern "C" {

/**
 * Gets some basic device information,
 * sets the device for the task,
 * and performs a simply alloc and transfer operation on GPU
 */
int horizontal_diffusion_gpu_init_(int *myproc, int *nproc, int *mydevice) {
	float x, *x_d;
	int i, dc;
	hipError_t cerr;
	char hostname[64];
	hipDeviceProp_t dp;

	hipEvent_t tS, tE;
	float timer = 0.0f;
	hipEventCreate(&tS);
	hipEventCreate(&tE);

	// Get some GPU device info
	hipGetDeviceCount(&dc);
	if (dc > 4) {
		fprintf(stderr, "Warning: more than %d devices on node (%d)\n", 4, dc);
		dc = 4;
	}

	fprintf(stderr, "Number of devices on this node: %d\n", dc);
	i = (*mydevice);
	if (dc > 0) {
		if ((cerr = hipSetDevice(i))) {
			fprintf(stderr, "Non-zero cerr %d\n", cerr);
		}
	}

	gethostname(hostname, 64);
	fprintf(stderr, "Setting device %02d for task %03d on host %s\n", i,
			*myproc, hostname);
	if ((cerr = hipGetDeviceProperties(&dp, i))) {
		fprintf(stderr, "Device %02d: cerr = %d\n", i, cerr);
	} else {
		fprintf(stderr, "Device %02d: name %s\n", i, dp.name);
		fprintf(stderr, "Device %02d: mem       %lu\n", i, dp.totalGlobalMem);
		fprintf(stderr, "Device %02d: smem      %lu\n", i,
				dp.sharedMemPerBlock);
		fprintf(stderr, "Device %02d: nreg      %d\n", i, dp.regsPerBlock);
		fprintf(stderr, "Device %02d: warp      %d\n", i, dp.warpSize);
		fprintf(stderr, "Device %02d: pitch     %lu\n", i, dp.memPitch);
		fprintf(stderr, "Device %02d: maxthrds  %d\n", i,
				dp.maxThreadsPerBlock);
		fprintf(stderr, "Device %02d: maxtdim   %d %d %d\n", i,
				(dp.maxThreadsDim)[0], (dp.maxThreadsDim)[1],
				(dp.maxThreadsDim)[2]);
		fprintf(stderr, "Device %02d: maxgdim   %d %d %d\n", i,
				(dp.maxGridSize)[0], (dp.maxGridSize)[1], (dp.maxGridSize)[2]);
		fprintf(stderr, "Device %02d: clock     %d\n", i, dp.clockRate);
		fprintf(stderr, "Device %02d: talign    %lu\n", i, dp.textureAlignment);
	}

	hipEventRecord(tS, NULL);

	hipMalloc((void **) (&x_d), sizeof(float));
	hipMemcpy(x_d, &x, sizeof(float), hipMemcpyHostToDevice);
	hipFree(x_d);

	hipEventRecord(tE, NULL);
	hipEventSynchronize(tE);
	hipEventElapsedTime(&timer, tS, tE);

	fprintf(stderr, "horizontal_diffusion_gpu_init: %.3f\n", timer);

	return 0;
}

/**
 * Convert fortran index to c index
 */
int indexI(int fi) {
	return fi + 4;
}
int indexJ(int fj) {
	return fj + 4;
}
int indexK(int fk) {
	return fk - 1;
}

// Dimensiones de las variables
int IMS = -4;
int IME = 430;
int JMS = -4;
int JME = 305;
int KMS = 1;
int KME = 35;
int IX = 435;
int JX = 310;
int KX = 35;

/**
 * Print variable to console
 */
void printVariable(const char name[], float *var, int ims, int ime, int kms,
		int kme, int jms, int jme) {
	printf("%s:\n", name);
	for (int k = indexK(kms); k <= indexK(kme); k++) {
		for (int j = indexJ(jms); j <= indexJ(jme); j++) {
			for (int i = indexI(ims); i <= indexI(ime); i++) {
				printf("%7.2f\t", var[i + k * IX + j * IX * KX]);
			}
			printf("\n");
		}
		printf("*\n");
	}
}

//[435][35][310]
int horizontal_diffusion_host_(int *ids, int *ide, int *jds, int *jde, int *kds,
		int *kde, int *ims, int *ime, int *jms, int *jme, int *kms, int *kme,
		int *its, int *ite, int *jts, int *jte, int *kts, int *kte,
		int *cf_specified, int *cf_nested, int *cf_open_xs, int *cf_open_xe,
		int *cf_open_ys, int *cf_open_ye, int *cf_periodic_x, int *cf_polar,
		char *name, float *field, float *tendency, float *mu, float *msfux,
		float *msfuy, float *msfvx, float *msfvx_inv, float *msfvy,
		float *msftx, float *msfty, float *khdif, float *xkmhd, float *rdx,
		float *rdy) {

	// Dimensions
	int d3 = (*ime - *ims + 1) * (*jme - *jms + 1) * (*kme - *kms + 1);
	int d2 = (*ime - *ims + 1) * (*jme - *jms + 1);

	// Timing data
	hipEvent_t tS0, tE0, tS1, tE1;
	hipEventCreate(&tS0);
	hipEventCreate(&tS1);
	hipEventCreate(&tE0);
	hipEventCreate(&tE1);
	float timer = 0.0f;

	printf("Dimensions:\n");
	printf("ids,ide,jds,jde,kds,kde: %d,%d,%d,%d,%d,%d\n", *ids, *ide, *jds,
			*jde, *kds, *kde);
	printf("ims,ime,jms,jme,kms,kme: %d,%d,%d,%d,%d,%d\n", *ims, *ime, *jms,
			*jme, *kms, *kme);
	printf("its,ite,jts,jte,kts,kte: %d,%d,%d,%d,%d,%d\n", *its, *ite, *jts,
			*jte, *kts, *kte);

	/*printf("Input variables: \n");
	 printf("Boolean: %d %d %d %d %d %d %d %d\n", *cf_specified, *cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye, *cf_periodic_x, *cf_polar);
	 printf("String: %c\n", *name);
	 printf("Float: %f %f %f\n", *khdif, *rdx, *rdy);*/

	// Starting transference of data to device memory
	hipEventRecord(tS0, NULL);

	TODEV3(field);
	TODEV3(tendency);
	TODEV3(xkmhd);
	TODEV2(mu);
	TODEV2(msfux);
	TODEV2(msfuy);
	TODEV2(msfvx);
	TODEV2(msfvx_inv);
	TODEV2(msfvy);
	TODEV2(msftx);
	TODEV2(msfty);

	// Main variable - before
	//printVariable("Tendency (input)", tendency, 100, 120, 1, 1, 200, 200);

	int remx, remy;
	remx = (*ime - *ims + 1) % XXX != 0 ? 1 : 0;
	remy = (*jme - *jms + 1) % YYY != 0 ? 1 : 0;

	dim3 dimBlock(XXX, YYY, 1);
	dim3 dimGrid(((*ime - *ims + 1) / XXX) + remx,
			((*jme - *jms + 1) / YYY) + remy, MKX);

	printf("Call to kernel: block dims %d %d %d\n", dimBlock.x, dimBlock.y,
			dimBlock.z);
	printf("Call to kernel: grid  dims %d %d %d\n", dimGrid.x, dimGrid.y,
			dimGrid.z);
	printf("Calling kernel \n");

	hipEventRecord(tS1, NULL);

	// Changing cache configuration
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(horizontal_diffusion_gpu), hipFuncCachePreferL1);

	// Set constant memory
	//hipMemcpyToSymbol(HIP_SYMBOL(&rdx_), &rdx, sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(&rdy_), &rdy, sizeof(float));

//for( int i=0; i<10; i++){

	hipError_t err = hipSuccess;
	horizontal_diffusion_gpu<<<dimGrid, dimBlock>>>(*ids, *ide, *jds, *jde,
			*kds, *kde, *ims, *ime, *jms, *jme, *kms, *kme, *its, *ite, *jts,
			*jte, *kts, *kte, *cf_specified, *cf_nested, *cf_open_xs,
			*cf_open_xe, *cf_open_ys, *cf_open_ye, *cf_periodic_x, *cf_polar,
			*name, field_d, tendency_d, mu_d, msfux_d, msfuy_d, msfvx_d,
			msfvx_inv_d, msfvy_d, msftx_d, msfty_d, *khdif, xkmhd_d, *rdx,
			*rdy);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit (EXIT_FAILURE);
	}

	hipDeviceSynchronize();

//}
	hipEventRecord(tE1, NULL);
	hipEventSynchronize(tE1);
	float timerRun;
	hipEventElapsedTime(&timerRun, tS1, tE1);

	fprintf(stderr, "Call to kernel (not including data xfer): %.3f msec\n",
			timerRun);
	//fprintf(stderr, "Call to kernel (not including data xfer): %.3f msec\n", timerRun/10);

	// Starting transference of output data from device
	FROMDEV3(tendency);

	hipEventRecord(tE0, NULL);
	hipEventSynchronize(tE0);
	hipEventElapsedTime(&timer, tS0, tE0);
	printf("Call to kernel (including data xfer): %.3f msec\n", timer);
	//printf("Call to kernel (including data xfer): %.3f msec\n", timer - timerRun + (timerRun/10));

	//printVariable("(hd.cu): Tendency (output)", tendency, 100, 120, 1, 1, 200, 200);

	CLNUP(field);
	CLNUP(tendency);
	CLNUP(xkmhd);
	CLNUP(mu);
	CLNUP(msfux);
	CLNUP(msfuy);
	CLNUP(msfvx);
	CLNUP(msfvx_inv);
	CLNUP(msfvy);
	CLNUP(msftx);
	CLNUP(msfty);

	return 0;
}

/*
 int get_horizontal_diffusion_gpu_levels(int *retval) {
 (*retval) = MKX;
 }
 */

}

