#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#include "hd_block_size.h"
#include "spt2.h"

__global__ void horizontal_diffusion_gpu(int ids, int ide, int jds, int jde,
		int kds, int kde, int ims, int ime, int jms, int jme, int kms, int kme,
		int its, int ite, int jts, int jte, int kts, int kte, int cf_specified,
		int cf_nested, int cf_open_xs, int cf_open_xe, int cf_open_ys,
		int cf_open_ye, int cf_periodic_x, int cf_polar, char name,
		float * field, float *tendency, float *mu, float *msfux, float *msfuy,
		float *msfvx, float *msfvx_inv, float *msfvy, float *msftx,
		float *msfty, float khdif, float *xkmhd, float rdx, float rdy) {

	// Local variables
	int i, j, k, ktf;
	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;

	// Vars in shared memory
	__shared__ float mu_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfux_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfuy_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msftx_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfty_s[((XXX + 2) * (YYY + 2))];

	if (cf_specified || cf_nested) {
		specified = 1;
	}

	ktf = min(kte, kde - 1);

	// Thread coordinates
	i = bi * bx + ti;
	j = bj * by + tj;

	if (name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 1, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = ite;

		// Debug info:

		// Check bounds before calculations!
		if (!(i >= i_start + 4 && i <= i_end + 4 && j >= j_start + 4
				&& j <= j_end + 4)) {
			return;
		}

		// Redundant copy to shared memory
		// The copy pattern:
		//  *   *
		//    t
		//  *   *
		mu_s[S2(ti-1,tj-1)] = mu[P2(ti-1,tj-1)];
		mu_s[S2(ti+1,tj-1)] = mu[P2(ti+1,tj-1)];
		mu_s[S2(ti-1,tj+1)] = mu[P2(ti-1,tj+1)];
		mu_s[S2(ti+1,tj+1)] = mu[P2(ti+1,tj+1)];

		/*msfux_s[S2(ti-1,tj-1)] = msfux[P2(ti-1,tj-1)];
		msfux_s[S2(ti+1,tj-1)] = msfux[P2(ti+1,tj-1)];
		msfux_s[S2(ti-1,tj+1)] = msfux[P2(ti-1,tj+1)];
		msfux_s[S2(ti+1,tj+1)] = msfux[P2(ti+1,tj+1)];*/
		msfux_s[S2(ti,tj-1)] = msfux[P2(ti,tj-1)];
		msfux_s[S2(ti,tj+1)] = msfux[P2(ti,tj+1)];

		/*msfuy_s[S2(ti-1,tj-1)] = msfuy[P2(ti-1,tj-1)];
		msfuy_s[S2(ti+1,tj-1)] = msfuy[P2(ti+1,tj-1)];
		msfuy_s[S2(ti-1,tj+1)] = msfuy[P2(ti-1,tj+1)];
		msfuy_s[S2(ti+1,tj+1)] = msfuy[P2(ti+1,tj+1)];*/
		msfuy_s[S2(ti,tj-1)] = msfuy[P2(ti,tj-1)];
		msfuy_s[S2(ti,tj+1)] = msfuy[P2(ti,tj+1)];

		/*msftx_s[S2(ti-1,tj-1)] = msftx[P2(ti-1,tj-1)];
		msftx_s[S2(ti+1,tj-1)] = msftx[P2(ti+1,tj-1)];
		msftx_s[S2(ti-1,tj+1)] = msftx[P2(ti-1,tj+1)];
		msftx_s[S2(ti+1,tj+1)] = msftx[P2(ti+1,tj+1)];*/
		msftx_s[S2(ti-1,tj)] = msftx[P2(ti-1,tj)];
		msftx_s[S2(ti+1,tj)] = msftx[P2(ti+1,tj)];
		
		/*msfty_s[S2(ti-1,tj-1)] = msfty[P2(ti-1,tj-1)];
		msfty_s[S2(ti+1,tj-1)] = msfty[P2(ti+1,tj-1)];
		msfty_s[S2(ti-1,tj+1)] = msfty[P2(ti-1,tj+1)];
		msfty_s[S2(ti+1,tj+1)] = msfty[P2(ti+1,tj+1)];*/
		msfty_s[S2(ti-1,tj)] = msfty[P2(ti-1,tj)];
		msfty_s[S2(ti+1,tj)] = msfty[P2(ti+1,tj)];

		__syncthreads();

		for (k = kts - 1; k <= ktf - 1; k++) {
			// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
			// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

			mkrdxm = (msftx_s[S2(ti-1,tj)] / msfty_s[S2(ti-1,tj)])
					* mu_s[S2(ti-1,tj)] * xkmhd[P3(ti-1,k,tj)] * rdx;
			mkrdxp = (msftx_s[S2(ti,tj)] / msfty_s[S2(ti,tj)]) * mu_s[S2(ti,tj)]
					* xkmhd[P3(ti,k,tj)] * rdx;
			mrdx = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * rdx;

			mkrdym = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj-1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj-1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj-1)] + mu_s[S2(ti-1,tj-1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* rdy;
			mkrdyp = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj+1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj+1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj+1)] + mu_s[S2(ti-1,tj+1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj+1)]
							+ xkmhd[P3(ti-1,k,tj+1)] + xkmhd[P3(ti-1,k,tj)])
					* rdy;

			// need to do four-corners (t) for diffusion coefficient as there are
			// no values at u,v points
			// msfuy - has to be y as part of d/dY
			//         has to be u as we're at a u point
			mrdy = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * rdy;

			// correctly averaged version of rho~ * m^2 *
			//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]
			/*tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
			 mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
			 -mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
			 +mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
			 -mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));*/
			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'v') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = jte;

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 1, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);
		if (cf_polar)
			j_start = max(jds + 1, jts);
		if (cf_polar)
			j_end = min(jde - 1, jte);

		// Check bounds before calculations!
		if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
				&& j <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = ((msfvx[P2(ti,tj)] + msfvx[P2(ti-1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti-1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti-1,tj-1)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* rdx;
			mkrdxp = ((msfvx[P2(ti,tj)] + msfvx[P2(ti+1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti+1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti+1,tj-1)]
							+ mu[P2(ti+1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti+1,k,tj-1)] + xkmhd[P3(ti+1,k,tj)])
					* rdx;
			mrdx = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdx;
			mkrdym = (msfty[P2(ti,tj-1)] / msftx[P2(ti,tj-1)])
					* xkmhd[P3(ti,k,tj-1)] * rdy;
			mkrdyp = (msfty[P2(ti,tj)] / msftx[P2(ti,tj)]) * xkmhd[P3(ti,k,tj)]
					* rdy;
			mrdy = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'w') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		// Check bounds before calculations!
		if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
				&& j <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)] + mu[P2(ti,tj)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti-1,k-1,tj)])
					* rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.25
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)] + mu[P2(ti+1,tj)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti+1,k-1,tj)] + xkmhd[P3(ti,k-1,tj)])
					* rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti,tj)]
							+ mu[P2(ti,tj-1)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti,k-1,tj-1)])
					* rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.25
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)] + mu[P2(ti,tj+1)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti,k-1,tj+1)] + xkmhd[P3(ti,k-1,tj)])
					* rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		for (k = kts; k <= ktf; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)]) * rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.5
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)]) * rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)]) * rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.5
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)]) * rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	}

}
