#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#include "hd_block_size.h"
#include "spt2.h"


__constant__ float rdx_[1];
__constant__ float rdy_[1];

__global__ void horizontal_diffusion_gpu(int ids, int ide, int jds, int jde,
		int kds, int kde, int ims, int ime, int jms, int jme, int kms, int kme,
		int its, int ite, int jts, int jte, int kts, int kte, int cf_specified,
		int cf_nested, int cf_open_xs, int cf_open_xe, int cf_open_ys,
		int cf_open_ye, int cf_periodic_x, int cf_polar, char name,
		float *field, float *tendency, float *mu, float *msfux, float *msfuy,
		float *msfvx, float *msfvx_inv, float *msfvy, float *msftx, float *msfty,
		float khdif, float *xkmhd, float rdx, float rdy){

	// Local variables
	int i, j, k, ktf;
	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;
	
	// Esto hace que aumente considerablemente el tiempo (+40%)
	// en lugar de disminuirlo

	__shared__ float xkmhd_s[((XXX+2)*(YYY+2)*(MKX))];

	if(cf_specified || cf_nested) {
		specified = 1;
	}

	ktf=min(kte,kde-1);

	// Thread coordinates
	i = bi*bx+ti;
	j = bj*by+tj;

	/*for( k=kts-1; k<=kte-1; k++) {
		if( !( i>=i_start-1 && i<=i_end-1 && j>=j_start-1 && j<=j_end-1 ) ){
			return;
		}
		tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+2;
	} return;*/

	if(name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = min(jte,jde-1);

		if( cf_open_xs || specified ) i_start = max(ids+1,its);
		if( cf_open_xe || specified ) i_end = min(ide-1,ite);
		if( cf_open_ys || specified ) j_start = max(jds+1,jts);
		if( cf_open_ye || specified ) j_end = min(jde-2,jte);
		if( cf_periodic_x ) i_start = its;
		if( cf_periodic_x ) i_end = ite;

		// Debug info:
		/*if(ti==0){
			tendency[0]=i_start;tendency[1]=i_end;tendency[2]=j_start;tendency[3]=j_end;
			tendency[4]=kts;tendency[5]=ktf;
		}*/

		// Check bounds before calculations!
		//if( !( i>=i_start-1 && i<=i_end-1 && j>=j_start-1 && j<=j_end-1 ) ){
		if( !( i>=i_start+4 && i<=i_end+4 && j>=j_start+4 && j<=j_end+4 ) ){
			// Debug
//			if(i<=ime-ims+1 && j<=jme-jms+1){
//				tendency[P3(ti,0,tj)] = -99.88;
//			}
			return;
		}
		
				// Redundant copy to shared memory
				for( k=kts-1; k<=ktf-1; k++) {
					xkmhd_s[S3(ti-1,k,tj-1)] = xkmhd[P3(ti-1,k,tj-1)];
					xkmhd_s[S3(ti+1,k,tj-1)] = xkmhd[P3(ti+1,k,tj-1)];
					xkmhd_s[S3(ti-1,k,tj+1)] = xkmhd[P3(ti-1,k,tj+1)];
					xkmhd_s[S3(ti+1,k,tj+1)] = xkmhd[P3(ti+1,k,tj+1)];
				}

				__syncthreads();


		for( k=kts-1; k<=ktf-1; k++) {
			// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
			// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

			mkrdxm=( msftx[P2(ti-1,tj)]/msfty[P2(ti-1,tj)] ) * mu[P2(ti-1,tj)] * xkmhd_s[S3(ti-1,k,tj)] * rdx;
			mkrdxp=( msftx[P2(ti,tj)]/msfty[P2(ti,tj)])*mu[P2(ti,tj)]*xkmhd_s[S3(ti,k,tj)]*rdx;
			mrdx=msfux[P2(ti,tj)]*msfuy[P2(ti,tj)]*rdx;

			mkrdym=( (msfuy[P2(ti,tj)]+msfuy[P2(ti,tj-1)])/(msfux[P2(ti,tj)]+msfux[P2(ti,tj-1)]) )*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)]+mu[P2(ti-1,tj-1)]+mu[P2(ti-1,tj)])*
					0.25*(xkmhd_s[S3(ti,k,tj)]+xkmhd_s[S3(ti,k,tj-1)]+xkmhd_s[S3(ti-1,k,tj-1)]+xkmhd_s[S3(ti-1,k,tj)])*rdy;
			mkrdyp=( (msfuy[P2(ti,tj)]+msfuy[P2(ti,tj+1)])/(msfux[P2(ti,tj)]+msfux[P2(ti,tj+1)]) )*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti,tj+1)]+mu[P2(ti-1,tj+1)]+mu[P2(ti-1,tj)])*
					0.25*(xkmhd_s[S3(ti,k,tj)]+xkmhd_s[S3(ti,k,tj+1)]+xkmhd_s[S3(ti-1,k,tj+1)]+xkmhd_s[S3(ti-1,k,tj)])*rdy;

			// need to do four-corners (t) for diffusion coefficient as there are
			// no values at u,v points
			// msfuy - has to be y as part of d/dY
			//         has to be u as we're at a u point
			mrdy=msfux[P2(ti,tj)]*msfuy[P2(ti,tj)]*rdy;

			// correctly averaged version of rho~ * m^2 *
			//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]
			/*tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
					mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
							-mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
							+mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
									-mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));*/
			tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
								mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
										-mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
										+mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
												-mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));
		}
	}else
		if(name == 'v') {

		i_start = its;
		i_end = min(ite,ide-1);
		j_start = jts;
		j_end = jte;

		if( cf_open_xs || specified ) i_start = max(ids+1,its);
		if( cf_open_xe || specified ) i_end = min(ide-2,ite);
		if( cf_open_ys || specified ) j_start = max(jds+1,jts);
		if( cf_open_ye || specified ) j_end = min(jde-1,jte);
		if( cf_periodic_x ) i_start = its;
		if( cf_periodic_x ) i_end = min(ite,ide-1);
		if( cf_polar ) j_start = max(jds+1,jts);
		if( cf_polar ) j_end = min(jde-1,jte);

		// Check bounds before calculations!
		if( !( i>=i_start-1 && i<=i_end-1 && j>=j_start-1 && j<=j_end-1 ) ){
			return;
		}

		for( k=kts-1; k<=ktf-1; k++ ) {

			mkrdxm=( (msfvx[P2(ti,tj)]+msfvx[P2(ti-1,tj)])/(msfvy[P2(ti,tj)]+msfvy[P2(ti-1,tj)]) )*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)]+mu[P2(ti-1,tj-1)]+mu[P2(ti-1,tj)])*
					0.25*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)]+xkmhd[P3(ti-1,k,tj-1)]+xkmhd[P3(ti-1,k,tj)])*rdx;
			mkrdxp=( (msfvx[P2(ti,tj)]+msfvx[P2(ti+1,tj)])/(msfvy[P2(ti,tj)]+msfvy[P2(ti+1,tj)]) )*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)]+mu[P2(ti+1,tj-1)]+mu[P2(ti+1,tj)])*
					0.25*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)]+xkmhd[P3(ti+1,k,tj-1)]+xkmhd[P3(ti+1,k,tj)])*rdx;
			mrdx=msfvx[P2(ti,tj)]*msfvy[P2(ti,tj)]*rdx;
			mkrdym=(msfty[P2(ti,tj-1)]/msftx[P2(ti,tj-1)])*xkmhd[P3(ti,k,tj-1)]*rdy;
			mkrdyp=(msfty[P2(ti,tj)]/msftx[P2(ti,tj)])*xkmhd[P3(ti,k,tj)]*rdy;
			mrdy=msfvx[P2(ti,tj)]*msfvy[P2(ti,tj)]*rdy;

			tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
					mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
							-mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
							+mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
									-mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));
		}
	}else if(name == 'w') {

		i_start = its;
		i_end = min(ite,ide-1);
		j_start = jts;
		j_end = min(jte,jde-1);

		if( cf_open_xs || specified ) i_start = max(ids+1,its);
		if( cf_open_xe || specified ) i_end = min(ide-2,ite);
		if( cf_open_ys || specified ) j_start = max(jds+1,jts);
		if( cf_open_ye || specified ) j_end = min(jde-2,jte);
		if( cf_periodic_x ) i_start = its;
		if( cf_periodic_x ) i_end = min(ite,ide-1);

		// Check bounds before calculations!
		if( !( i>=i_start-1 && i<=i_end-1 && j>=j_start-1 && j<=j_end-1 ) ){
			return;
		}

		for( k=kts-1; k<=ktf-1; k++ ) {

			mkrdxm=(msfux[P2(ti,tj)]/msfuy[P2(ti,tj)])*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti-1,tj)]+mu[P2(ti,tj)]+mu[P2(ti-1,tj)])*
					0.25*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti-1,k,tj)]+xkmhd[P3(ti,k-1,tj)]+xkmhd[P3(ti-1,k-1,tj)])*rdx;
			mkrdxp=(msfux[P2(ti+1,tj)]/msfuy[P2(ti+1,tj)])*
					0.25*(mu[P2(ti+1,tj)]+mu[P2(ti,tj)]+mu[P2(ti+1,tj)]+mu[P2(ti,tj)])*
					0.25*(xkmhd[P3(ti+1,k,tj)]+xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti+1,k-1,tj)]+xkmhd[P3(ti,k-1,tj)])*rdx;
			mrdx=msftx[P2(ti,tj)]*msfty[P2(ti,tj)]*rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*
			mkrdym=(msfvy[P2(ti,tj)]*msfvx_inv[P2(ti,tj)])*
					0.25*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)]+mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*
					0.25*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)]+xkmhd[P3(ti,k-1,tj)]+xkmhd[P3(ti,k-1,tj-1)])*rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*
			mkrdyp=(msfvy[P2(ti,tj+1)]*msfvx_inv[P2(ti,tj+1)])*
					0.25*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)]+mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*
					0.25*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k-1,tj+1)]+xkmhd[P3(ti,k-1,tj)])*rdy;
			mrdy=msftx[P2(ti,tj)]*msfty[P2(ti,tj)]*rdy;

			tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
					mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
							-mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
							+mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
									-mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));
		}
	}else {

		i_start = its;
		i_end = min(ite,ide-1);
		j_start = jts;
		j_end = min(jte,jde-1);

		if( cf_open_xs || specified ) i_start = max(ids+1,its);
		if( cf_open_xe || specified ) i_end = min(ide-2,ite);
		if( cf_open_ys || specified ) j_start = max(jds+1,jts);
		if( cf_open_ye || specified ) j_end = min(jde-2,jte);
		if( cf_periodic_x ) i_start = its;
		if( cf_periodic_x ) i_end = min(ite,ide-1);

		for( k=kts; k<=ktf; k++ ) {

			mkrdxm=(msfux[P2(ti,tj)]/msfuy[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti-1,k,tj)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti-1,tj)])*rdx;
			mkrdxp=(msfux[P2(ti+1,tj)]/msfuy[P2(ti+1,tj)])*0.5*(xkmhd[P3(ti+1,k,tj)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti+1,tj)]+mu[P2(ti,tj)])*rdx;
			mrdx=msftx[P2(ti,tj)]*msfty[P2(ti,tj)]*rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy
			mkrdym=(msfvy[P2(ti,tj)]*msfvx_inv[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy
			mkrdyp=(msfvy[P2(ti,tj+1)]*msfvx_inv[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy;
			mrdy=msftx[P2(ti,tj)]*msfty[P2(ti,tj)]*rdy;

			tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
					mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
							-mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
							+mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
									-mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));
		}
	}

}
