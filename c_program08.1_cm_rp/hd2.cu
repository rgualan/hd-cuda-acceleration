#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>
#include "hd_block_size.h"
#include "spt2.h"

//#include <stdio.h>
//#include "hd_block_size.h"

#define TODEV(A,s) float *A##_d;hipMalloc((void**)&A##_d,((s))*sizeof(float));hipMemcpy(A##_d,A,(s)*sizeof(float),hipMemcpyHostToDevice);
#define FROMDEV(A,s) hipMemcpy(A,A##_d,(s)*sizeof(float),hipMemcpyDeviceToHost);
#define CLNUP(A) hipFree(A##_d)

#define TODEV3(A) TODEV(A,d3)
#define TODEV2(A) TODEV(A,d2)
#define FROMDEV3(A) FROMDEV(A,d3)
#define FROMDEV2(A) FROMDEV(A,d2)

extern "C" int gethostname(char * name, size_t len);

//extern __device__ __constant__ float rdx_;
//extern __device__ __constant__ float rdy_;
__constant__ float rdx_c[1];
__constant__ float rdy_c[1];

__global__ void horizontal_diffusion_gpu(int ids, int ide, int jds, int jde,
		int kds, int kde, int ims, int ime, int jms, int jme, int kms, int kme,
		int its, int ite, int jts, int jte, int kts, int kte, int cf_specified,
		int cf_nested, int cf_open_xs, int cf_open_xe, int cf_open_ys,
		int cf_open_ye, int cf_periodic_x, int cf_polar, char name,
		float * __restrict__ field, float * __restrict__ tendency,
		float * __restrict__ mu, float * __restrict__ msfux,
		float * __restrict__ msfuy, float * __restrict__ msfvx,
		float * __restrict__ msfvx_inv, float * __restrict__ msfvy,
		float * __restrict__ msftx, float * __restrict__ msfty, float khdif,
		float * __restrict__ xkmhd, float rdx, float rdy) {

	// Local variables
	//int i, j, k, ktf;
	int k, ktf;
	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;

	// Vars in shared memory
	__shared__ float mu_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfux_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfuy_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msftx_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfty_s[((XXX + 2) * (YYY + 2))];

	if (cf_specified || cf_nested) {
		specified = 1;
	}

	ktf = min(kte, kde - 1);

	if (name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 1, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = ite;

		// Check bounds before calculations!
		if (!(ii >= i_start + 4 && ij <= i_end + 4 && ij >= j_start + 4
				&& ij <= j_end + 4)) {
			return;
		}

		// Redundant copy to shared memory
		mu_s[S2(ti-1,tj-1)] = mu[P2(ti-1,tj-1)];
		mu_s[S2(ti+1,tj-1)] = mu[P2(ti+1,tj-1)];
		mu_s[S2(ti-1,tj+1)] = mu[P2(ti-1,tj+1)];
		mu_s[S2(ti+1,tj+1)] = mu[P2(ti+1,tj+1)];

		msfux_s[S2(ti-1,tj-1)] = msfux[P2(ti-1,tj-1)];
		msfux_s[S2(ti+1,tj-1)] = msfux[P2(ti+1,tj-1)];
		msfux_s[S2(ti-1,tj+1)] = msfux[P2(ti-1,tj+1)];
		msfux_s[S2(ti+1,tj+1)] = msfux[P2(ti+1,tj+1)];

		msfuy_s[S2(ti-1,tj-1)] = msfuy[P2(ti-1,tj-1)];
		msfuy_s[S2(ti+1,tj-1)] = msfuy[P2(ti+1,tj-1)];
		msfuy_s[S2(ti-1,tj+1)] = msfuy[P2(ti-1,tj+1)];
		msfuy_s[S2(ti+1,tj+1)] = msfuy[P2(ti+1,tj+1)];

		msftx_s[S2(ti-1,tj-1)] = msftx[P2(ti-1,tj-1)];
		msftx_s[S2(ti+1,tj-1)] = msftx[P2(ti+1,tj-1)];
		msftx_s[S2(ti-1,tj+1)] = msftx[P2(ti-1,tj+1)];
		msftx_s[S2(ti+1,tj+1)] = msftx[P2(ti+1,tj+1)];

		msfty_s[S2(ti-1,tj-1)] = msfty[P2(ti-1,tj-1)];
		msfty_s[S2(ti+1,tj-1)] = msfty[P2(ti+1,tj-1)];
		msfty_s[S2(ti-1,tj+1)] = msfty[P2(ti-1,tj+1)];
		msfty_s[S2(ti+1,tj+1)] = msfty[P2(ti+1,tj+1)];

		__syncthreads();

		//printf("rdx: %f, rdy: %f \n", rdx_c[0], rdy_c[0]);

		for (k = kts - 1; k <= ktf - 1; k++) {
			// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
			// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

			mkrdxm = (msftx_s[S2(ti-1,tj)] / msfty_s[S2(ti-1,tj)])
					* mu_s[S2(ti-1,tj)] * xkmhd[P3(ti-1,k,tj)] * (rdx_c[0]);
			mkrdxp = (msftx_s[S2(ti,tj)] / msfty_s[S2(ti,tj)]) * mu_s[S2(ti,tj)]
					* xkmhd[P3(ti,k,tj)] * (rdx_c[0]);
			mrdx = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * (rdx_c[0]);

			mkrdym = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj-1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj-1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj-1)] + mu_s[S2(ti-1,tj-1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* (rdy_c[0]);
			mkrdyp = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj+1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj+1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj+1)] + mu_s[S2(ti-1,tj+1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj+1)]
							+ xkmhd[P3(ti-1,k,tj+1)] + xkmhd[P3(ti-1,k,tj)])
					* (rdy_c[0]);

			// need to do four-corners (t) for diffusion coefficient as there are
			// no values at u,v points
			// msfuy - has to be y as part of d/dY
			//         has to be u as we're at a u point
			mrdy = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * (rdy_c[0]);

			// correctly averaged version of rho~ * m^2 *
			//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]
			/*tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
			 mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
			 -mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
			 +mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
			 -mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));*/
			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'v') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = jte;

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 1, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);
		if (cf_polar)
			j_start = max(jds + 1, jts);
		if (cf_polar)
			j_end = min(jde - 1, jte);

		// Check bounds before calculations!
		if (!(ii >= i_start - 1 && ii <= i_end - 1 && ij >= j_start - 1
				&& ij <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = ((msfvx[P2(ti,tj)] + msfvx[P2(ti-1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti-1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti-1,tj-1)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* rdx;
			mkrdxp = ((msfvx[P2(ti,tj)] + msfvx[P2(ti+1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti+1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti+1,tj-1)]
							+ mu[P2(ti+1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti+1,k,tj-1)] + xkmhd[P3(ti+1,k,tj)])
					* rdx;
			mrdx = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdx;
			mkrdym = (msfty[P2(ti,tj-1)] / msftx[P2(ti,tj-1)])
					* xkmhd[P3(ti,k,tj-1)] * rdy;
			mkrdyp = (msfty[P2(ti,tj)] / msftx[P2(ti,tj)]) * xkmhd[P3(ti,k,tj)]
					* rdy;
			mrdy = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'w') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		// Check bounds before calculations!
		if (!(ii >= i_start - 1 && ii <= i_end - 1 && ij >= j_start - 1
				&& ij <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)] + mu[P2(ti,tj)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti-1,k-1,tj)])
					* rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.25
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)] + mu[P2(ti+1,tj)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti+1,k-1,tj)] + xkmhd[P3(ti,k-1,tj)])
					* rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti,tj)]
							+ mu[P2(ti,tj-1)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti,k-1,tj-1)])
					* rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.25
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)] + mu[P2(ti,tj+1)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti,k-1,tj+1)] + xkmhd[P3(ti,k-1,tj)])
					* rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		for (k = kts; k <= ktf; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)]) * rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.5
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)]) * rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)]) * rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.5
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)]) * rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	}

}

extern "C" {

/**
 * Gets some basic device information,
 * sets the device for the task,
 * and performs a simply alloc and transfer operation on GPU
 */
int horizontal_diffusion_gpu_init_(int *myproc, int *nproc, int *mydevice) {
	float x, *x_d;
	int i, dc;
	hipError_t cerr;
	char hostname[64];
	hipDeviceProp_t dp;

	hipEvent_t tS, tE;
	float timer = 0.0f;
	hipEventCreate(&tS);
	hipEventCreate(&tE);

	// Get some GPU device info
	hipGetDeviceCount(&dc);
	if (dc > 4) {
		fprintf(stderr, "Warning: more than %d devices on node (%d)\n", 4, dc);
		dc = 4;
	}

	fprintf(stderr, "Number of devices on this node: %d\n", dc);
	i = (*mydevice);
	if (dc > 0) {
		if ((cerr = hipSetDevice(i))) {
			fprintf(stderr, "Non-zero cerr %d\n", cerr);
		}
	}

	gethostname(hostname, 64);
	fprintf(stderr, "Setting device %02d for task %03d on host %s\n", i,
			*myproc, hostname);
	if ((cerr = hipGetDeviceProperties(&dp, i))) {
		fprintf(stderr, "Device %02d: cerr = %d\n", i, cerr);
	} else {
		fprintf(stderr, "Device %02d: name %s\n", i, dp.name);
		fprintf(stderr, "Device %02d: mem       %lu\n", i, dp.totalGlobalMem);
		fprintf(stderr, "Device %02d: smem      %lu\n", i,
				dp.sharedMemPerBlock);
		fprintf(stderr, "Device %02d: nreg      %d\n", i, dp.regsPerBlock);
		fprintf(stderr, "Device %02d: warp      %d\n", i, dp.warpSize);
		fprintf(stderr, "Device %02d: pitch     %lu\n", i, dp.memPitch);
		fprintf(stderr, "Device %02d: maxthrds  %d\n", i,
				dp.maxThreadsPerBlock);
		fprintf(stderr, "Device %02d: maxtdim   %d %d %d\n", i,
				(dp.maxThreadsDim)[0], (dp.maxThreadsDim)[1],
				(dp.maxThreadsDim)[2]);
		fprintf(stderr, "Device %02d: maxgdim   %d %d %d\n", i,
				(dp.maxGridSize)[0], (dp.maxGridSize)[1], (dp.maxGridSize)[2]);
		fprintf(stderr, "Device %02d: clock     %d\n", i, dp.clockRate);
		fprintf(stderr, "Device %02d: talign    %lu\n", i, dp.textureAlignment);
	}

	hipEventRecord(tS, NULL);

	hipMalloc((void **) (&x_d), sizeof(float));
	hipMemcpy(x_d, &x, sizeof(float), hipMemcpyHostToDevice);
	hipFree(x_d);

	hipEventRecord(tE, NULL);
	hipEventSynchronize(tE);
	hipEventElapsedTime(&timer, tS, tE);

	fprintf(stderr, "horizontal_diffusion_gpu_init: %.3f\n", timer);

	return 0;
}

/**
 * Convert fortran index to c index
 */
int indexI(int fi) {
	return fi + 4;
}
int indexJ(int fj) {
	return fj + 4;
}
int indexK(int fk) {
	return fk - 1;
}

// Dimensiones de las variables
int IMS = -4;
int IME = 430;
int JMS = -4;
int JME = 305;
int KMS = 1;
int KME = 35;
int IX = 435;
int JX = 310;
int KX = 35;

/**
 * Print variable to console
 */
void printVariable(const char name[], float *var, int ims, int ime, int kms,
		int kme, int jms, int jme) {
	printf("%s:\n", name);
	for (int k = indexK(kms); k <= indexK(kme); k++) {
		for (int j = indexJ(jms); j <= indexJ(jme); j++) {
			for (int i = indexI(ims); i <= indexI(ime); i++) {
				printf("%7.2f\t", var[i + k * IX + j * IX * KX]);
			}
			printf("\n");
		}
		printf("*\n");
	}
}

//[435][35][310]
int horizontal_diffusion_host_(int *ids, int *ide, int *jds, int *jde, int *kds,
		int *kde, int *ims, int *ime, int *jms, int *jme, int *kms, int *kme,
		int *its, int *ite, int *jts, int *jte, int *kts, int *kte,
		int *cf_specified, int *cf_nested, int *cf_open_xs, int *cf_open_xe,
		int *cf_open_ys, int *cf_open_ye, int *cf_periodic_x, int *cf_polar,
		char *name, float *field, float *tendency, float *mu, float *msfux,
		float *msfuy, float *msfvx, float *msfvx_inv, float *msfvy,
		float *msftx, float *msfty, float *khdif, float *xkmhd, float *rdx,
		float *rdy) {

	// Dimensions
	int d3 = (*ime - *ims + 1) * (*jme - *jms + 1) * (*kme - *kms + 1);
	int d2 = (*ime - *ims + 1) * (*jme - *jms + 1);

	// Timing data
	hipEvent_t tS0, tE0, tS1, tE1;
	hipEventCreate(&tS0);
	hipEventCreate(&tS1);
	hipEventCreate(&tE0);
	hipEventCreate(&tE1);
	float timer = 0.0f;

	printf("Dimensions:\n");
	printf("ids,ide,jds,jde,kds,kde: %d,%d,%d,%d,%d,%d\n", *ids, *ide, *jds,
			*jde, *kds, *kde);
	printf("ims,ime,jms,jme,kms,kme: %d,%d,%d,%d,%d,%d\n", *ims, *ime, *jms,
			*jme, *kms, *kme);
	printf("its,ite,jts,jte,kts,kte: %d,%d,%d,%d,%d,%d\n", *its, *ite, *jts,
			*jte, *kts, *kte);

	/*printf("Input variables: \n");
	 printf("Boolean: %d %d %d %d %d %d %d %d\n", *cf_specified, *cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye, *cf_periodic_x, *cf_polar);
	 printf("String: %c\n", *name);
	 printf("Float: %f %f %f\n", *khdif, *rdx, *rdy);*/

	// Starting transference of data to device memory
	hipEventRecord(tS0, NULL);

	TODEV3(field);
	TODEV3(tendency);
	TODEV3(xkmhd);
	TODEV2(mu);
	TODEV2(msfux);
	TODEV2(msfuy);
	TODEV2(msfvx);
	TODEV2(msfvx_inv);
	TODEV2(msfvy);
	TODEV2(msftx);
	TODEV2(msfty);

	// Main variable - before
	//printVariable("Tendency (input)", tendency, 100, 120, 1, 1, 200, 200);

	int remx, remy;
	remx = (*ime - *ims + 1) % XXX != 0 ? 1 : 0;
	remy = (*jme - *jms + 1) % YYY != 0 ? 1 : 0;

	dim3 dimBlock(XXX, YYY);
	dim3 dimGrid(((*ime - *ims + 1) / XXX) + remx,
			((*jme - *jms + 1) / YYY) + remy);

	printf("Call to kernel: block dims %d %d\n", dimBlock.x, dimBlock.y);
	printf("Call to kernel: grid  dims %d %d\n", dimGrid.x, dimGrid.y);
	printf("Calling kernel \n");

	hipEventRecord(tS1, NULL);

	// Set constant memory
	//hipMemcpyToSymbol(HIP_SYMBOL(rdx_), rdx, sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(rdy_), rdy, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(rdx_c), rdx, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rdy_c), rdy, sizeof(float));

	// Changing cache configuration
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(horizontal_diffusion_gpu), hipFuncCachePreferL1);

//for( int i=0; i<10; i++){

	hipError_t err = hipSuccess;
	horizontal_diffusion_gpu<<<dimGrid, dimBlock>>>(*ids, *ide, *jds, *jde,
			*kds, *kde, *ims, *ime, *jms, *jme, *kms, *kme, *its, *ite, *jts,
			*jte, *kts, *kte, *cf_specified, *cf_nested, *cf_open_xs,
			*cf_open_xe, *cf_open_ys, *cf_open_ye, *cf_periodic_x, *cf_polar,
			*name, field_d, tendency_d, mu_d, msfux_d, msfuy_d, msfvx_d,
			msfvx_inv_d, msfvy_d, msftx_d, msfty_d, *khdif, xkmhd_d, *rdx,
			*rdy);
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit (EXIT_FAILURE);
	}

	hipDeviceSynchronize();

//}
	hipEventRecord(tE1, NULL);
	hipEventSynchronize(tE1);
	float timerRun;
	hipEventElapsedTime(&timerRun, tS1, tE1);

	fprintf(stderr, "Call to kernel (not including data xfer): %.3f msec\n",
			timerRun);
	//fprintf(stderr, "Call to kernel (not including data xfer): %.3f msec\n", timerRun/10);

	// Starting transference of output data from device
	FROMDEV3(tendency);

	hipEventRecord(tE0, NULL);
	hipEventSynchronize(tE0);
	hipEventElapsedTime(&timer, tS0, tE0);
	printf("Call to kernel (including data xfer): %.3f msec\n", timer);
	//printf("Call to kernel (including data xfer): %.3f msec\n", timer - timerRun + (timerRun/10));

	//printVariable("(hd.cu): Tendency (output)", tendency, 100, 120, 1, 1, 200, 200);

	CLNUP(field);
	CLNUP(tendency);
	CLNUP(xkmhd);
	CLNUP(mu);
	CLNUP(msfux);
	CLNUP(msfuy);
	CLNUP(msfvx);
	CLNUP(msfvx_inv);
	CLNUP(msfvy);
	CLNUP(msftx);
	CLNUP(msfty);

	return 0;
}

}
