#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hd_block_size.h"

#define TODEV(A,s) float *A##_d;hipMalloc((void**)&A##_d,((s))*sizeof(float));hipMemcpy(A##_d,A,(s)*sizeof(float),hipMemcpyHostToDevice);
#define FROMDEV(A,s) hipMemcpy(A,A##_d,(s)*sizeof(float),hipMemcpyDeviceToHost);
#define CLNUP(A) hipFree(A##_d)

#define TODEV3(A) TODEV(A,d3)
#define TODEV2(A) TODEV(A,d2)
#define FROMDEV3(A) FROMDEV(A,d3)
#define FROMDEV2(A) FROMDEV(A,d2)

/*extern __global__ void horizontal_diffusion_gpu(int ids, int ide, int jds,
		int jde, int kds, int kde, int ims, int ime, int jms, int jme, int kms,
		int kme, int its, int ite, int jts, int jte, int kts, int kte,
		int cf_specified, int cf_nested, int cf_open_xs, int cf_open_xe,
		int cf_open_ys, int cf_open_ye, int cf_periodic_x, int cf_polar,
		char name, float *field, float *tendency, float *mu, float *msfux,
		float *msfuy, float *msfvx, float *msfvx_inv, float *msfvy,
		float *msftx, float *msfty, float khdif, float *xkmhd, float rdx,
		float rdy);*/

__global__ void horizontal_diffusion_gpu_u(int i_start, int i_end, int j_start,
		int j_end, int k_start, int k_end, int ims, int ime, int jms, int jme,
		int kms, int kme, float * field, float *tendency, float *mu,
		float *msfux, float *msfuy, float *msfvx, float *msfvx_inv,
		float *msfvy, float *msftx, float *msfty, float khdif, float * xkmhd,
		float rdx, float rdy);

extern "C" int gethostname(char * name, size_t len);

extern "C" {

/**
 * Gets some basic device information,
 * sets the device for the task,
 * and performs a simply alloc and transfer operation on GPU
 */
int horizontal_diffusion_gpu_init_(int *myproc, int *nproc, int *mydevice) {
	float x, *x_d;
	int i, dc;
	hipError_t cerr;
	char hostname[64];
	hipDeviceProp_t dp;

	hipEvent_t tS, tE;
	float timer = 0.0f;
	hipEventCreate(&tS);
	hipEventCreate(&tE);

	// Get some GPU device info
	hipGetDeviceCount(&dc);
	if (dc > 4) {
		fprintf(stderr, "Warning: more than %d devices on node (%d)\n", 4, dc);
		dc = 4;
	}

	fprintf(stderr, "Number of devices on this node: %d\n", dc);
	i = (*mydevice);
	if (dc > 0) {
		if ((cerr = hipSetDevice(i))) {
			fprintf(stderr, "Non-zero cerr %d\n", cerr);
		}
	}

	gethostname(hostname, 64);
	fprintf(stderr, "Setting device %02d for task %03d on host %s\n", i,
			*myproc, hostname);
	if ((cerr = hipGetDeviceProperties(&dp, i))) {
		fprintf(stderr, "Device %02d: cerr = %d\n", i, cerr);
	} else {
		fprintf(stderr, "Device %02d: name %s\n", i, dp.name);
		fprintf(stderr, "Device %02d: mem       %lu\n", i, dp.totalGlobalMem);
		fprintf(stderr, "Device %02d: smem      %lu\n", i,
				dp.sharedMemPerBlock);
		fprintf(stderr, "Device %02d: nreg      %d\n", i, dp.regsPerBlock);
		fprintf(stderr, "Device %02d: warp      %d\n", i, dp.warpSize);
		fprintf(stderr, "Device %02d: pitch     %lu\n", i, dp.memPitch);
		fprintf(stderr, "Device %02d: maxthrds  %d\n", i,
				dp.maxThreadsPerBlock);
		fprintf(stderr, "Device %02d: maxtdim   %d %d %d\n", i,
				(dp.maxThreadsDim)[0], (dp.maxThreadsDim)[1],
				(dp.maxThreadsDim)[2]);
		fprintf(stderr, "Device %02d: maxgdim   %d %d %d\n", i,
				(dp.maxGridSize)[0], (dp.maxGridSize)[1], (dp.maxGridSize)[2]);
		fprintf(stderr, "Device %02d: clock     %d\n", i, dp.clockRate);
		fprintf(stderr, "Device %02d: talign    %lu\n", i, dp.textureAlignment);
	}

	hipEventRecord(tS, NULL);

	hipMalloc((void **) (&x_d), sizeof(float));
	hipMemcpy(x_d, &x, sizeof(float), hipMemcpyHostToDevice);
	hipFree(x_d);

	hipEventRecord(tE, NULL);
	hipEventSynchronize(tE);
	hipEventElapsedTime(&timer, tS, tE);

	fprintf(stderr, "horizontal_diffusion_gpu_init: %.3f\n", timer);

	return 0;
}

/**
 * Convert fortran index to c index
 */
int indexI(int fi) {
	return fi + 4;
}
int indexJ(int fj) {
	return fj + 4;
}
int indexK(int fk) {
	return fk - 1;
}

// Dimensiones de las variables
int IMS = -4;
int IME = 430;
int JMS = -4;
int JME = 305;
int KMS = 1;
int KME = 35;
int IX = 435;
int JX = 310;
int KX = 35;

/**
 * Print variable to console
 */
void printVariable(const char name[], float *var, int ims, int ime, int kms,
		int kme, int jms, int jme) {
	printf("%s:\n", name);
	for (int k = indexK(kms); k <= indexK(kme); k++) {
		for (int j = indexJ(jms); j <= indexJ(jme); j++) {
			for (int i = indexI(ims); i <= indexI(ime); i++) {
				printf("%7.2f\t", var[i + k * IX + j * IX * KX]);
			}
			printf("\n");
		}
		printf("*\n");
	}
}

void horizontal_diffusion_pre(int ids, int ide, int jds, int jde, int kds,
		int kde, int ims, int ime, int jms, int jme, int kms, int kme, int its,
		int ite, int jts, int jte, int kts, int kte, int cf_specified,
		int cf_nested, int cf_open_xs, int cf_open_xe, int cf_open_ys,
		int cf_open_ye, int cf_periodic_x, int cf_polar, char name,
		float *field, float *tendency, float *mu, float *msfux, float *msfuy,
		float *msfvx, float *msfvx_inv, float *msfvy, float *msftx,
		float *msfty, float khdif, float *xkmhd, float rdx, float rdy) {

	// Local variables
	int ktf;
	int i_start, i_end, j_start, j_end;
	int specified = 0;
	// Timing data
	hipEvent_t tS0, tE0, tS1, tE1;
	hipEventCreate(&tS0);
	hipEventCreate(&tS1);
	hipEventCreate(&tE0);
	hipEventCreate(&tE1);
	float timer = 0.0f;


	// Host-device transfer
	// Dimensions
	int d3 = (ime - ims + 1) * (jme - jms + 1) * (kme - kms + 1);
	int d2 = (ime - ims + 1) * (jme - jms + 1);

	// Starting transference of data to device memory
	hipEventRecord(tS0, NULL);
	TODEV3(field);
	TODEV3(tendency);
	TODEV3(xkmhd);
	TODEV2(mu);
	TODEV2(msfux);
	TODEV2(msfuy);
	TODEV2(msfvx);
	TODEV2(msfvx_inv);
	TODEV2(msfvy);
	TODEV2(msftx);
	TODEV2(msfty);

	// Overall process
	if (cf_specified || cf_nested) {
		specified = 1;
	}

	ktf = min(kte, kde - 1);

	if (name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 1, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = ite;

		// Use i_start, i_end, j_start, j_end to calculate a fit grid/block for GPU computation
		printf("Call to kernel: is, ie, js, je:  %d %d %d %d\n", i_start, i_end, j_start, j_end);

		int remx, remy;
		remx = (i_end - i_start + 1) % XXX != 0 ? 1 : 0;
		remy = (j_end - j_start + 1) % YYY != 0 ? 1 : 0;


		dim3 dimBlock(XXX, YYY);
		dim3 dimGrid(((i_end - i_start + 1) / XXX) + remx,
				((j_end - j_start + 1) / YYY) + remy); // (+4-4)

		printf("Call to kernel: block dims %d %d\n", dimBlock.x, dimBlock.y);
		printf("Call to kernel: grid  dims %d %d\n", dimGrid.x, dimGrid.y);
		printf("Calling kernel-u \n");

		hipEventRecord(tS1, NULL);

		// Changing cache configuration
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(horizontal_diffusion_gpu_u), hipFuncCachePreferL1);


		horizontal_diffusion_gpu_u<<<dimGrid, dimBlock>>>(i_start+4, i_end+4, j_start+4, j_end+4, kts, ktf,
				ims, ime, jms, jme, kms, kme,
				field_d, tendency_d, mu_d, msfux_d, msfuy_d, msfvx_d, msfvx_inv_d,
				msfvy_d, msftx_d, msfty_d, khdif, xkmhd_d, rdx, rdy);
		hipDeviceSynchronize();

	} else if (name == 'v') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = jte;

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 1, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);
		if (cf_polar)
			j_start = max(jds + 1, jts);
		if (cf_polar)
			j_end = min(jde - 1, jte);

		// Call kernel-v

	} else if (name == 'w') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		//call kernel-w

	} else {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		//call kernel-other
	}

	hipEventRecord(tE1, NULL);
	hipEventSynchronize(tE1);
	float timerRun;
	hipEventElapsedTime(&timerRun, tS1, tE1);

	fprintf(stderr, "Call to kernel (not including data xfer): %.3f msec\n", timerRun);


	// Device-host transfer
	FROMDEV3(tendency);

	hipEventRecord(tE0, NULL);
	hipEventSynchronize(tE0);
	hipEventElapsedTime(&timer, tS0, tE0);
	printf("Call to kernel (including data xfer): %.3f msec\n", timer);

	// Clean device variables
	CLNUP(field);
	CLNUP(tendency);
	CLNUP(xkmhd);
	CLNUP(mu);
	CLNUP(msfux);
	CLNUP(msfuy);
	CLNUP(msfvx);
	CLNUP(msfvx_inv);
	CLNUP(msfvy);
	CLNUP(msftx);
	CLNUP(msfty);

}

//[435][35][310]
int horizontal_diffusion_host_(int *ids, int *ide, int *jds, int *jde, int *kds,
		int *kde, int *ims, int *ime, int *jms, int *jme, int *kms, int *kme,
		int *its, int *ite, int *jts, int *jte, int *kts, int *kte,
		int *cf_specified, int *cf_nested, int *cf_open_xs, int *cf_open_xe,
		int *cf_open_ys, int *cf_open_ye, int *cf_periodic_x, int *cf_polar,
		char *name, float *field, float *tendency, float *mu, float *msfux,
		float *msfuy, float *msfvx, float *msfvx_inv, float *msfvy,
		float *msftx, float *msfty, float *khdif, float *xkmhd, float *rdx,
		float *rdy) {

	/*printf("Dimensions:\n");
	printf("ids,ide,jds,jde,kds,kde: %d,%d,%d,%d,%d,%d\n", *ids, *ide, *jds,
			*jde, *kds, *kde);
	printf("ims,ime,jms,jme,kms,kme: %d,%d,%d,%d,%d,%d\n", *ims, *ime, *jms,
			*jme, *kms, *kme);
	printf("its,ite,jts,jte,kts,kte: %d,%d,%d,%d,%d,%d\n", *its, *ite, *jts,
			*jte, *kts, *kte);

	printf("Input variables: \n");
	printf("Boolean: %d %d %d %d %d %d %d %d\n", *cf_specified, *cf_nested,
			*cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye, *cf_periodic_x,
			*cf_polar);
	printf("String: %c\n", *name);
	printf("Float: %f %f %f\n", *khdif, *rdx, *rdy);*/


	// Main variable - before
	//printVariable("Tendency (input)", tendency, 100, 120, 1, 1, 200, 200);



//for( int i=0; i<10; i++){

	/*horizontal_diffusion_gpu<<<dimGrid, dimBlock>>>(*ids, *ide, *jds, *jde,
	 *kds, *kde, *ims, *ime, *jms, *jme, *kms, *kme,
	 *its, *ite, *jts, *jte, *kts, *kte, *cf_specified,
	 *cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys,
	 *cf_open_ye, *cf_periodic_x, *cf_polar, *name,
	 field_d, tendency_d, mu_d, msfux_d, msfuy_d,
	 msfvx_d, msfvx_inv_d, msfvy_d, msftx_d, msfty_d,
	 *khdif, xkmhd_d, *rdx, *rdy);*/
	horizontal_diffusion_pre(*ids, *ide, *jds, *jde, *kds, *kde, *ims, *ime,
			*jms, *jme, *kms, *kme, *its, *ite, *jts, *jte, *kts, *kte,
			*cf_specified, *cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys,
			*cf_open_ye, *cf_periodic_x, *cf_polar, *name, field, tendency,
			mu, msfux, msfuy, msfvx, msfvx_inv, msfvy, msftx,
			msfty, *khdif, xkmhd, *rdx, *rdy);

//}


	//printVariable("(hd.cu): Tendency (output)", tendency, 100, 120, 1, 1, 200, 200);

	return 0;
}

/*
 int get_horizontal_diffusion_gpu_levels(int *retval) {
 (*retval) = MKX;
 }
 */

}

