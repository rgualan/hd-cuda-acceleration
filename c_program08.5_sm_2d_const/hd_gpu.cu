#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>

#include "hd_block_size.h"
#include "spt2.h"

__global__ void horizontal_diffusion_gpu(const int ids, const int ide,
		const int jds, const int jde, const int kds, const int kde,
		const int ims, const int ime, const int jms, const int jme,
		const int kms, const int kme, const int its, const int ite,
		const int jts, const int jte, const int kts, const int kte,
		const int cf_specified, const int cf_nested, const int cf_open_xs,
		const int cf_open_xe, const int cf_open_ys, const int cf_open_ye,
		const int cf_periodic_x, const int cf_polar, const char name,
		const float * __restrict__ field, float * __restrict__ tendency, const float * __restrict__ mu,
		const float * __restrict__ msfux, const float * __restrict__ msfuy, const float * __restrict__ msfvx,
		const float * __restrict__ msfvx_inv, const float * __restrict__ msfvy, const float * __restrict__ msftx,
		const float * __restrict__ msfty, const float khdif, const float * __restrict__ xkmhd,
		const float rdx, const float rdy) {
	// Local variables
	//int i, j, k, ktf;
	int k;
	int i_start, i_end, j_start, j_end;
	float mrdx, mkrdxm, mkrdxp, mrdy, mkrdym, mkrdyp;
	int specified = 0;

	// Vars in shared memory
	__shared__ float mu_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfux_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfuy_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msftx_s[((XXX + 2) * (YYY + 2))];
	__shared__ float msfty_s[((XXX + 2) * (YYY + 2))];

	if (cf_specified || cf_nested) {
		specified = 1;
	}

	const int ktf = min(kte, kde - 1);

	// Thread coordinates
	const int i = bi * bx + ti;
	const int j = bj * by + tj;

	if (name == 'u') {
		i_start = its;
		i_end = ite;
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 1, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = ite;

		// Debug info:

		// Check bounds before calculations!
		if (!(i >= i_start + 4 && i <= i_end + 4 && j >= j_start + 4
				&& j <= j_end + 4)) {
			return;
		}

		// Redundant copy to shared memory
		// The copy pattern:
		//  *   *
		//    t
		//  *   *
		mu_s[S2(ti-1,tj-1)] = mu[P2(ti-1,tj-1)];
		mu_s[S2(ti+1,tj-1)] = mu[P2(ti+1,tj-1)];
		mu_s[S2(ti-1,tj+1)] = mu[P2(ti-1,tj+1)];
		mu_s[S2(ti+1,tj+1)] = mu[P2(ti+1,tj+1)];

		msfux_s[S2(ti-1,tj-1)] = msfux[P2(ti-1,tj-1)];
		msfux_s[S2(ti+1,tj-1)] = msfux[P2(ti+1,tj-1)];
		msfux_s[S2(ti-1,tj+1)] = msfux[P2(ti-1,tj+1)];
		msfux_s[S2(ti+1,tj+1)] = msfux[P2(ti+1,tj+1)];

		msfuy_s[S2(ti-1,tj-1)] = msfuy[P2(ti-1,tj-1)];
		msfuy_s[S2(ti+1,tj-1)] = msfuy[P2(ti+1,tj-1)];
		msfuy_s[S2(ti-1,tj+1)] = msfuy[P2(ti-1,tj+1)];
		msfuy_s[S2(ti+1,tj+1)] = msfuy[P2(ti+1,tj+1)];

		msftx_s[S2(ti-1,tj-1)] = msftx[P2(ti-1,tj-1)];
		msftx_s[S2(ti+1,tj-1)] = msftx[P2(ti+1,tj-1)];
		msftx_s[S2(ti-1,tj+1)] = msftx[P2(ti-1,tj+1)];
		msftx_s[S2(ti+1,tj+1)] = msftx[P2(ti+1,tj+1)];

		msfty_s[S2(ti-1,tj-1)] = msfty[P2(ti-1,tj-1)];
		msfty_s[S2(ti+1,tj-1)] = msfty[P2(ti+1,tj-1)];
		msfty_s[S2(ti-1,tj+1)] = msfty[P2(ti-1,tj+1)];
		msfty_s[S2(ti+1,tj+1)] = msfty[P2(ti+1,tj+1)];

		__syncthreads();

		for (k = kts - 1; k <= ktf - 1; k++) {
			// The interior is grad: (m_x*d/dx), the exterior is div: (m_x*m_y*d/dx(/m_y))
			// setting up different averagings of m^2 partial d/dX and m^2 partial d/dY

			mkrdxm = (msftx_s[S2(ti-1,tj)] / msfty_s[S2(ti-1,tj)])
					* mu_s[S2(ti-1,tj)] * xkmhd[P3(ti-1,k,tj)] * rdx;
			mkrdxp = (msftx_s[S2(ti,tj)] / msfty_s[S2(ti,tj)]) * mu_s[S2(ti,tj)]
					* xkmhd[P3(ti,k,tj)] * rdx;
			mrdx = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * rdx;

			mkrdym = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj-1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj-1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj-1)] + mu_s[S2(ti-1,tj-1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* rdy;
			mkrdyp = ((msfuy_s[S2(ti,tj)] + msfuy_s[S2(ti,tj+1)])
					/ (msfux_s[S2(ti,tj)] + msfux_s[S2(ti,tj+1)])) * 0.25
					* (mu_s[S2(ti,tj)] + mu_s[S2(ti,tj+1)] + mu_s[S2(ti-1,tj+1)]
							+ mu_s[S2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj+1)]
							+ xkmhd[P3(ti-1,k,tj+1)] + xkmhd[P3(ti-1,k,tj)])
					* rdy;

			// need to do four-corners (t) for diffusion coefficient as there are
			// no values at u,v points
			// msfuy - has to be y as part of d/dY
			//         has to be u as we're at a u point
			mrdy = msfux_s[S2(ti,tj)] * msfuy_s[S2(ti,tj)] * rdy;

			// correctly averaged version of rho~ * m^2 *
			//    [partial d/dX(partial du^/dX) + partial d/dY(partial du^/dY)]
			/*tendency[P3(ti,k,tj)]=tendency[P3(ti,k,tj)]+(
			 mrdx*(mkrdxp*(field[P3(ti+1,k,tj)]-field[P3(ti,k,tj)])
			 -mkrdxm*(field[P3(ti,k,tj)]-field[P3(ti-1,k,tj)]))
			 +mrdy*(mkrdyp*(field[P3(ti,k,tj+1)]-field[P3(ti,k,tj)])
			 -mkrdym*(field[P3(ti,k,tj)]-field[P3(ti,k,tj-1)])));*/
			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'v') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = jte;

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 1, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);
		if (cf_polar)
			j_start = max(jds + 1, jts);
		if (cf_polar)
			j_end = min(jde - 1, jte);

		// Check bounds before calculations!
		if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
				&& j <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = ((msfvx[P2(ti,tj)] + msfvx[P2(ti-1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti-1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti-1,tj-1)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti-1,k,tj-1)] + xkmhd[P3(ti-1,k,tj)])
					* rdx;
			mkrdxp = ((msfvx[P2(ti,tj)] + msfvx[P2(ti+1,tj)])
					/ (msfvy[P2(ti,tj)] + msfvy[P2(ti+1,tj)])) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti+1,tj-1)]
							+ mu[P2(ti+1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti+1,k,tj-1)] + xkmhd[P3(ti+1,k,tj)])
					* rdx;
			mrdx = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdx;
			mkrdym = (msfty[P2(ti,tj-1)] / msftx[P2(ti,tj-1)])
					* xkmhd[P3(ti,k,tj-1)] * rdy;
			mkrdyp = (msfty[P2(ti,tj)] / msftx[P2(ti,tj)]) * xkmhd[P3(ti,k,tj)]
					* rdy;
			mrdy = msfvx[P2(ti,tj)] * msfvy[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else if (name == 'w') {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		// Check bounds before calculations!
		if (!(i >= i_start - 1 && i <= i_end - 1 && j >= j_start - 1
				&& j <= j_end - 1)) {
			return;
		}

		for (k = kts - 1; k <= ktf - 1; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)] + mu[P2(ti,tj)]
							+ mu[P2(ti-1,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti-1,k-1,tj)])
					* rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.25
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)] + mu[P2(ti+1,tj)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti+1,k-1,tj)] + xkmhd[P3(ti,k-1,tj)])
					* rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.25
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)] + mu[P2(ti,tj)]
							+ mu[P2(ti,tj-1)]) * 0.25
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]
							+ xkmhd[P3(ti,k-1,tj)] + xkmhd[P3(ti,k-1,tj-1)])
					* rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.25
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)] + mu[P2(ti,tj+1)]
							+ mu[P2(ti,tj)]) * 0.25
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]
							+ xkmhd[P3(ti,k-1,tj+1)] + xkmhd[P3(ti,k-1,tj)])
					* rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	} else {

		i_start = its;
		i_end = min(ite, ide - 1);
		j_start = jts;
		j_end = min(jte, jde - 1);

		if (cf_open_xs || specified)
			i_start = max(ids + 1, its);
		if (cf_open_xe || specified)
			i_end = min(ide - 2, ite);
		if (cf_open_ys || specified)
			j_start = max(jds + 1, jts);
		if (cf_open_ye || specified)
			j_end = min(jde - 2, jte);
		if (cf_periodic_x)
			i_start = its;
		if (cf_periodic_x)
			i_end = min(ite, ide - 1);

		for (k = kts; k <= ktf; k++) {

			mkrdxm = (msfux[P2(ti,tj)] / msfuy[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti-1,k,tj)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti-1,tj)]) * rdx;
			mkrdxp = (msfux[P2(ti+1,tj)] / msfuy[P2(ti+1,tj)]) * 0.5
					* (xkmhd[P3(ti+1,k,tj)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti+1,tj)] + mu[P2(ti,tj)]) * rdx;
			mrdx = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdx;
			//         mkrdym=(msfvy[P2(ti,tj)]/msfvx[P2(ti,tj)])*0.5*(xkmhd[P3(ti,k,tj)]+xkmhd[P3(ti,k,tj-1)])*0.5*(mu[P2(ti,tj)]+mu[P2(ti,tj-1)])*rdy
			mkrdym = (msfvy[P2(ti,tj)] * msfvx_inv[P2(ti,tj)]) * 0.5
					* (xkmhd[P3(ti,k,tj)] + xkmhd[P3(ti,k,tj-1)]) * 0.5
					* (mu[P2(ti,tj)] + mu[P2(ti,tj-1)]) * rdy;
			//         mkrdyp=(msfvy[P2(ti,tj+1)]/msfvx[P2(ti,tj+1)])*0.5*(xkmhd[P3(ti,k,tj+1)]+xkmhd[P3(ti,k,tj)])*0.5*(mu[P2(ti,tj+1)]+mu[P2(ti,tj)])*rdy
			mkrdyp = (msfvy[P2(ti,tj+1)] * msfvx_inv[P2(ti,tj+1)]) * 0.5
					* (xkmhd[P3(ti,k,tj+1)] + xkmhd[P3(ti,k,tj)]) * 0.5
					* (mu[P2(ti,tj+1)] + mu[P2(ti,tj)]) * rdy;
			mrdy = msftx[P2(ti,tj)] * msfty[P2(ti,tj)] * rdy;

			tendency[P3(ti,k,tj)] =
					tendency[P3(ti,k,tj)]
							+ (mrdx
									* (mkrdxp
											* (field[P3(ti+1,k,tj)]
													- field[P3(ti,k,tj)])
											- mkrdxm
													* (field[P3(ti,k,tj)]
															- field[P3(ti-1,k,tj)]))
									+ mrdy
											* (mkrdyp
													* (field[P3(ti,k,tj+1)]
															- field[P3(ti,k,tj)])
													- mkrdym
															* (field[P3(ti,k,tj)]
																	- field[P3(ti,k,tj-1)])));
		}
	}

}
