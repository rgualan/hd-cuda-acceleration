#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h> // CUDA runtime
#include <hip/hip_runtime_api.h>  // Utilities to work with CUDA
#include "spt1.h"
#include "multiGPU.h"
#include "hd_block_size.h"
#include "util.h"

/////////////////////////////////////
// Functional macros
// Moving data
// To host (page-lock/pinned)
//#define TOHOST(A,d,s) checkCudaErrors(hipHostMalloc((void**)&plan[i].A##_h,(d)*sizeof(float)));memcpy(&plan[i].A##_h[0], &A[(s)], (d)*sizeof(float))
#define TOHOST(A,d,s) hipHostMalloc((void**)&plan[i].A##_h,(d)*sizeof(float));memcpy(&plan[i].A##_h[0], &A[(s)], (d)*sizeof(float))
#define TOHOST3(A) TOHOST(A,plan[i].d3,plan[i].s3)
#define TOHOST2(A) TOHOST(A,plan[i].d2,plan[i].s2)

// To device, from pinned memory
// Part 1: allocation
#define ALLOCD(A,d) hipMalloc((void**)&plan[i].A##_d,(d)*sizeof(float))
#define ALLOCD3(A) ALLOCD(A,plan[i].d3)
#define ALLOCD2(A) ALLOCD(A,plan[i].d2)

// Part 2: async transfer
//#define TODEVUPM(A,d) printf("TODEV:%d\n",d);checkCudaErrors(hipMalloc((void**)&plan[i].A##_d,(d)*sizeof(float)));printf("Pasa hipMalloc\n");checkCudaErrors(hipMemcpyAsync(plan[i].A##_d, plan[i].A##_h, (d)*sizeof(float),hipMemcpyHostToDevice, plan[i].stream));printf("Pasa hipMemcpyAsync\n");
//#define TODEV(A,d) hipMalloc((void**)&plan[i].A##_d,(d)*sizeof(float));hipMemcpyAsync(plan[i].A##_d, plan[i].A##_h, (d)*sizeof(float),hipMemcpyHostToDevice, plan[i].stream)
#define TODEV(A,d) hipMemcpyAsync(plan[i].A##_d, plan[i].A##_h, (d)*sizeof(float),hipMemcpyHostToDevice, plan[i].stream)
#define TODEV3(A) TODEV(A,plan[i].d3)
#define TODEV2(A) TODEV(A,plan[i].d2)

// From host - to device, using pinned memory
#define TODEVUPM3(A) TOHOST3(A);TODEV3(A)
#define TODEVUPM2(A) TOHOST2(A);TODEV2(A)

// To host, from device, through pinned memory
//#define FROMDEV(A,d1,d2,s1,s2) checkCudaErrors(hipMemcpyAsync(plan[i].A##_h,plan[i].A##_d,(d1)*sizeof(float),hipMemcpyDeviceToHost,plan[i].stream));memcpy(&A[s1], &plan[i].A##_h[(s2)], (d2)*sizeof(float));
//#define FROMDEV(A,d1,d2,s1,s2) hipMemcpyAsync(&plan[i].A##_h, &plan[i].A##_d, (d1) *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream);memcpy(&A[s1], &plan[i].A##_h[(s2)], (d2)*sizeof(float));
//#define FROMDEV3(A) FROMDEV(A,plan[i].d3,plan[i].d3_,plan[i].s3_,plan[i].delta)
//#define FROMDEV2(A) FROMDEV(A,plan[i].d2,plan[i].d2_,plan[i].s2_,plan[i].delta)

#define CLNUP(A) hipFree(plan[i].A##_d);hipHostFree(plan[i].A##_h);

/////////////////////////////////////
// Constants
#define MAX_GPU_COUNT 32 // Max number of GPUs used
#define STREAMS_N 2 // Streams per GPU
/////////////////////////////////////

// Methods
extern __global__ void horizontal_diffusion_kernel(int ids, int ide, int jds, int jde, int kds,
		int kde, int ims, int ime, int jms, int jme, int kms, int kme, int its, int ite, int jts,
		int jte, int kts, int kte, int cf_specified, int cf_nested, int cf_open_xs, int cf_open_xe,
		int cf_open_ys, int cf_open_ye, int cf_periodic_x, int cf_polar, char name, float *field,
		float *tendency, float *mu, float *msfux, float *msfuy, float *msfvx, float *msfvx_inv,
		float *msfvy, float *msftx, float *msfty, float khdif, float *xkmhd, float rdx, float rdy,
		int sub_js, int sub_je, int data_js, int data_je);

extern "C" {

int activeGpu = -1;
void setActiveGpu(int i) {
	if (activeGpu != i) {
		activeGpu = i;
		printf("Active GPU: %d\n", activeGpu);
		checkCudaErrors((hipSetDevice(activeGpu)));
	}
}

int horizontal_diffusion_host(int *ids, int *ide, int *jds, int *jde, int *kds, int *kde, int *ims,
		int *ime, int *jms, int *jme, int *kms, int *kme, int *its, int *ite, int *jts, int *jte,
		int *kts, int *kte, int *cf_specified, int *cf_nested, int *cf_open_xs, int *cf_open_xe,
		int *cf_open_ys, int *cf_open_ye, int *cf_periodic_x, int *cf_polar, char *name,
		float *field, float *tendency, float *mu, float *msfux, float *msfuy, float *msfvx,
		float *msfvx_inv, float *msfvy, float *msftx, float *msfty, float *khdif, float *xkmhd,
		float *rdx, float *rdy) {

	int g, s, i; // Index
	int GPU_N; // Number of GPUs

	// Original dimensions (Not considering sub-division in j dimension) (For indexing macros)
	const int ix = *ime - *ims + 1;
	const int kx = *kme - *kms + 1;

	printf("Input name: %c\n", *name);
	printf("Tendency first element: %f\n", tendency[P3(6,0,6)]);

	printf("Dimensions:\n");
	printf("ids,ide,jds,jde,kds,kde: %4d,%4d,%4d,%4d,%4d,%4d\n", *ids, *ide, *jds, *jde, *kds,
			*kde);
	printf("ims,ime,jms,jme,kms,kme: %4d,%4d,%4d,%4d,%4d,%4d\n", *ims, *ime, *jms, *jme, *kms,
			*kme);
	printf("its,ite,jts,jte,kts,kte: %4d,%4d,%4d,%4d,%4d,%4d\n", *its, *ite, *jts, *jte, *kts,
			*kte);

	// Solver configuration
	// This structure has the basic I/O data for each calculation of a kernel
	TGPUplan plan[MAX_GPU_COUNT * STREAMS_N];

	// Get the number of available devices in the node
	// The limit is MAX_GPU_COUNT
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	if (GPU_N > MAX_GPU_COUNT) {
		GPU_N = MAX_GPU_COUNT;
	}
	printf("CUDA-capable device count: %i\n", GPU_N);

	// Subdivide workload magnitude
	printf("Subdividing workload...\n");
	int tasks = GPU_N * STREAMS_N;
	i = 0;
	for (g = 0; g < GPU_N; g++) {
		for (s = 0; s < STREAMS_N; s++) {
			plan[i].gpu_id = g;
			plan[i].sub_dim_j = (*jme - *jms + 1) / tasks;
			i++;
		}
	}

	//Take into account "odd" data sizes
	for (i = 0; i < (*jme - *jms + 1) % tasks; i++) {
		plan[i].sub_dim_j++;
	}

	// Define indices for processing and for sub-data loading
	for (i = 0; i < tasks; i++) {
		// Sub-region processing indices
		if (i == 0) {
			plan[i].sub_js = *jms;
			plan[i].sub_je = plan[i].sub_js + plan[i].sub_dim_j - 1;
		} else {
			plan[i].sub_js = plan[i - 1].sub_je + 1;
			plan[i].sub_je = plan[i].sub_js + plan[i].sub_dim_j - 1;
		}
		printf("Subdivision of proc (GPU %d-%d) in j dim: (%4d,%4d) (%d)\n", plan[i].gpu_id, i,
				plan[i].sub_js, plan[i].sub_je, plan[i].sub_dim_j);

		// Sub-region data indices
		if (plan[i].sub_js - 1 < (*ims))
			plan[i].data_js = (*ims);
		else
			plan[i].data_js = plan[i].sub_js - 1;

		if (plan[i].sub_je + 1 > (*jme))
			plan[i].data_je = (*jme);
		else
			plan[i].data_je = plan[i].sub_je + 1;
		printf("Subdivision of data (GPU %d-%d) in j dim: (%4d,%4d) (%d)\n", plan[i].gpu_id, i,
				plan[i].data_js, plan[i].data_je, (plan[i].data_je - plan[i].data_js + 1));

		// Indices and dimensions for transference
		plan[i].s3 = J(plan[i].data_js) * ((*ime - *ims + 1) * (*kme - *kms + 1));
		plan[i].s2 = J(plan[i].data_js) * ((*ime - *ims + 1));
		plan[i].d3 = (*ime - *ims + 1) * (*kme - *kms + 1)
				* (plan[i].data_je - plan[i].data_js + 1);
		plan[i].d2 = (*ime - *ims + 1) * (plan[i].data_je - plan[i].data_js + 1);

		// Indices for returning data
		plan[i].s3_ = J(plan[i].sub_js) * ((*ime - *ims + 1) * (*kme - *kms + 1));
		plan[i].d3_ = (*ime - *ims + 1) * (*kme - *kms + 1) * (plan[i].sub_je - plan[i].sub_js + 1);
		plan[i].delta = (plan[i].sub_js - plan[i].data_js) * (*ime - *ims + 1) * (*kme - *kms + 1);

	}

	// Do memory allocation before asynchronous tasks
	for (i = 0; i < tasks; i++) {
		setActiveGpu(plan[i].gpu_id);

		startTimer(2);
		TOHOST3(field);
		TOHOST3(tendency);
		TOHOST3(xkmhd);
		TOHOST2(mu);
		TOHOST2(msfux);
		TOHOST2(msfuy);
		TOHOST2(msfvx);
		TOHOST2(msfvx_inv);
		TOHOST2(msfvy);
		TOHOST2(msftx);
		TOHOST2(msfty);
		stopTimer(2);
		printf("Time for duplicate data from normal memory to pinned memory: %d ms.\n",
				getElapsedTimeMs(2));

		startTimer(2);
		ALLOCD3(field);
		ALLOCD3(tendency);
		ALLOCD3(xkmhd);
		ALLOCD2(mu);
		ALLOCD2(msfux);
		ALLOCD2(msfuy);
		ALLOCD2(msfvx);
		ALLOCD2(msfvx_inv);
		ALLOCD2(msfvy);
		ALLOCD2(msftx);
		ALLOCD2(msfty);
		stopTimer(2);
		printf("Time for device memory allocation: %d ms.\n", getElapsedTimeMs(2));
	}

	//Start timer
	printf("Computing with %d GPUs and %d streams\n", GPU_N, STREAMS_N);
	startTimer(1);

	//Create streams for issuing GPU command asynchronously and allocate memory
	for (i = 0; i < tasks; i++) {
		setActiveGpu(plan[i].gpu_id);
		// Stream
		plan[i].stream = (hipStream_t) malloc(sizeof(hipStream_t));
		checkCudaErrors(hipStreamCreate(&plan[i].stream));
	}

	// a:1) (tx,proc,tx), ...
	// b:0) (tx,tx,...), (proc,proc,...), (tx,tx,...), ...
#if 0
	// Running a)
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		//Transfer macros to host
		/*
		 startTimer(2);
		 TOHOST3(field);
		 TOHOST3(tendency);
		 TOHOST3(xkmhd);
		 TOHOST2(mu);
		 TOHOST2(msfux);
		 TOHOST2(msfuy);
		 TOHOST2(msfvx);
		 TOHOST2(msfvx_inv);
		 TOHOST2(msfvy);
		 TOHOST2(msftx);
		 TOHOST2(msfty);
		 stopTimer(2);
		 printf("Time for duplicate data from normal memory to pinned memory: %d ms.\n",
		 getElapsedTimeMs(2));*/

		// Transfer macros to device
		TODEV3(field);
		TODEV3(tendency);
		TODEV3(xkmhd);
		TODEV2(mu);
		TODEV2(msfux);
		TODEV2(msfuy);
		TODEV2(msfvx);
		TODEV2(msfvx_inv);
		TODEV2(msfvy);
		TODEV2(msftx);
		TODEV2(msfty);

		/*//Transfer macros to device through pinned memory
		 TODEVUPM3(field); TODEVUPM3(tendency); TODEVUPM3(xkmhd); TODEVUPM2(mu); TODEVUPM2(msfux); TODEVUPM2(msfuy); TODEVUPM2(msfvx); TODEVUPM2(msfvx_inv); TODEVUPM2(msfvy); TODEVUPM2(msftx); TODEVUPM2(msfty);*/

		//printf("Computing with %d-%d (gpu-stream)...\n", plan[i].gpu_id, i);
		int remx, remy;
		remx = (*ime - *ims + 1) % XXX != 0 ? 1 : 0;
		remy = (plan[i].sub_je - plan[i].sub_js + 1) % YYY != 0 ? 1 : 0;
		dim3 dimBlock(XXX, YYY);
		dim3 dimGrid(((*ime - *ims + 1) / XXX) + remx,
				((plan[i].sub_je - plan[i].sub_js + 1) / YYY) + remy);

		//printf("Block dims (%4d,%4d)\n", dimBlock.x, dimBlock.y);
		//printf("Grid  dims (%4d,%4d)\n", dimGrid.x, dimGrid.y);

		horizontal_diffusion_kernel<<<dimGrid, dimBlock, 0, plan[i].stream>>>(
				*ids, *ide, *jds, *jde, *kds, *kde, *ims, *ime, *jms,
				*jme, *kms, *kme, *its, *ite, *jts, *jte, *kts, *kte, *cf_specified,
				*cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye,
				*cf_periodic_x, *cf_polar, *name, plan[i].field_d, plan[i].tendency_d, plan[i].mu_d,
				plan[i].msfux_d, plan[i].msfuy_d, plan[i].msfvx_d, plan[i].msfvx_inv_d, plan[i].msfvy_d, plan[i].msftx_d,
				plan[i].msfty_d, *khdif, plan[i].xkmhd_d, *rdx, *rdy, plan[i].sub_js,
				plan[i].sub_je, plan[i].data_js, plan[i].data_je);
		//getLastCudaError("<Kernel execution failed>");

		//Read back GPU results to pinned memory (Part 1)
		hipMemcpyAsync(plan[i].tendency_h, plan[i].tendency_d, (plan[i].d3) * sizeof(float),
				hipMemcpyDeviceToHost, plan[i].stream);
	}

#else
	// Running b)

	// Tx to device
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		/*
		 //Transfer macros to host
		 TOHOST3(field);
		 TOHOST3(tendency);
		 TOHOST3(xkmhd);
		 TOHOST2(mu);
		 TOHOST2(msfux);
		 TOHOST2(msfuy);
		 TOHOST2(msfvx);
		 TOHOST2(msfvx_inv);
		 TOHOST2(msfvy);
		 TOHOST2(msftx);
		 TOHOST2(msfty);*/

		// Transfer macros to device
		TODEV3(field);
		TODEV3(tendency);
		TODEV3(xkmhd);
		TODEV2(mu);
		TODEV2(msfux);
		TODEV2(msfuy);
		TODEV2(msfvx);
		TODEV2(msfvx_inv);
		TODEV2(msfvy);
		TODEV2(msftx);
		TODEV2(msfty);
	}

	// Computing
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		printf("Computing with %d-%d (gpu-stream)...\n", plan[i].gpu_id, i);

		int remx, remy;
		remx = (*ime - *ims + 1) % XXX != 0 ? 1 : 0;
		remy = (plan[i].sub_je - plan[i].sub_js + 1) % YYY != 0 ? 1 : 0;
		dim3 dimBlock(XXX, YYY);
		dim3 dimGrid(((*ime - *ims + 1) / XXX) + remx,
				((plan[i].sub_je - plan[i].sub_js + 1) / YYY) + remy);

		printf("Block dims (%4d,%4d)\n", dimBlock.x, dimBlock.y);
		printf("Grid  dims (%4d,%4d)\n", dimGrid.x, dimGrid.y);

		horizontal_diffusion_kernel<<<dimGrid, dimBlock, 0, plan[i].stream>>>(
				*ids, *ide, *jds, *jde, *kds, *kde, *ims, *ime, *jms,
				*jme, *kms, *kme, *its, *ite, *jts, *jte, *kts, *kte, *cf_specified,
				*cf_nested, *cf_open_xs, *cf_open_xe, *cf_open_ys, *cf_open_ye,
				*cf_periodic_x, *cf_polar, *name, plan[i].field_d, plan[i].tendency_d, plan[i].mu_d,
				plan[i].msfux_d, plan[i].msfuy_d, plan[i].msfvx_d, plan[i].msfvx_inv_d, plan[i].msfvy_d, plan[i].msftx_d,
				plan[i].msfty_d, *khdif, plan[i].xkmhd_d, *rdx, *rdy, plan[i].sub_js,
				plan[i].sub_je, plan[i].data_js, plan[i].data_je);
		//getLastCudaError("<Kernel execution failed>");

	}

	// Tx from device
	// Computing
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		//Read back GPU results to pinned memory (Part 1)
		hipMemcpyAsync(plan[i].tendency_h, plan[i].tendency_d, (plan[i].d3) * sizeof(float),
				hipMemcpyDeviceToHost, plan[i].stream);
	}
#endif

	//Synchronize GPU, retrieve data (Part 2), and shutdown
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		//Wait for all operations to finish
		hipStreamSynchronize(plan[i].stream);
	}

	//Synchronize GPU, retrieve data (Part 2), and shutdown
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		//Wait for all operations to finish
		//hipStreamSynchronize(plan[i].stream);

		// Read back GPU result to Host memory from pinned memory
		memcpy(&tendency[plan[i].s3_], &plan[i].tendency_h[(plan[i].delta)],
				(plan[i].d3_) * sizeof(float));
	}

	// Stop timer
	stopTimer(1);
	printf("Runtime for executing the kernel using streams: %d ms.\n", getElapsedTimeMs(1));

	//Shut down the GPU
	for (i = 0; i < tasks; i++) {
		//Set active GPU
		setActiveGpu(plan[i].gpu_id);

		// Clean memory
		startTimer(2);
		CLNUP(field);
		CLNUP(tendency);
		CLNUP(xkmhd);
		CLNUP(mu);
		CLNUP(msfux);
		CLNUP(msfuy);
		CLNUP(msfvx);
		CLNUP(msfvx_inv);
		CLNUP(msfvy);
		CLNUP(msftx);
		CLNUP(msfty);
		stopTimer(2);
		printf("Time for cleaning up: %d ms.\n", getElapsedTimeMs(2));

		hipStreamDestroy(plan[i].stream);
		//checkCudaErrors(hipDeviceReset());
		//sleep(1);
	}

	//Device reset
	for (i = 0; i < GPU_N; i++) {
		//Set active GPU
		printf("Reseting device %d\n", i);
		setActiveGpu(i);
		hipDeviceReset();
	}

	return 0;
}

}

